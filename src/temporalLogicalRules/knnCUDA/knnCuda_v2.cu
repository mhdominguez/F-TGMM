#include "hip/hip_runtime.h"
/*
 * knnCuda.cu
 *
 *  Created on: Jul 15, 2011
 *      Author: amatf
 */

#include "knnCuda_v2.h"
#include "book.h"
#include <algorithm>

#if defined(_WIN32) || defined(_WIN64)
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#endif


//static const int MAX_REF_POINTS=5000;//we need to predefined this in order to store reference points as constant memory. Total memory needed is MAX_QUERY_POINTS*3*4 bytes. It can not be more than 5400!!!

#ifndef CUDA_MAX_SIZE_CONST //to protect agains teh same constant define in other places in the code
#define CUDA_MAX_SIZE_CONST
    #ifndef CUDA_CONSTANTS_FA
    #define CUDA_CONSTANTS_FA
        static const int MAX_THREADS=1024;//For Quadro4800->256;//to make sure we don't run out of registers; For TeslaC2070 -> 1024
        //static const int MAX_THREADS_CUDA=1024;//For Quadro4800->512;//certain kernels benefit from maximum number of threads
        static const int MAX_BLOCKS=65535;
    #endif
#endif



#ifndef DIMS_IMAGE_CONST //to protect agains teh same constant define in other places in the code
#define DIMS_IMAGE_CONST
static const int dimsImage = 3;//to be able to precompile code
#endif


__constant__ float knn_v2_scaleCUDA[dimsImage];


__device__ inline void findMaxPosition(float *distArray,float* minDist,int *pos, int KNN)
{
	(*minDist)=distArray[0];
	(*pos)=0;
	for(int ii=1;ii<KNN;ii++) 
	{
		if((*minDist)<distArray[ii])
		{
			(*minDist)=distArray[ii];
			(*pos)=ii;
		}
	}
}


//===========================================================================================
__global__ void __launch_bounds__(MAX_THREADS) knnKernelNoConstantMemory(int *indCUDA,float *distCUDA,float *queryCUDA,float* anchorCUDA,int ref_nb,long long int query_nb, int KNN)
{
	// map from threadIdx/BlockIdx to pixel position
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int offset = blockDim.x * gridDim.x;


	if(KNN > knn_v2_maxKNN) return;//code is not ready for this
	if(tid>=query_nb) return;

	
	//int kMinusOne=knn_v2_maxKNN-1;
	float minDist[knn_v2_maxKNN];//to mantain distance for each index: since K is very small instead of a priority queue we keep a sorted array
	int indAux[knn_v2_maxKNN];
	float queryAux[dimsImage];
	float minDistThr;
	
	float dist,distAux;
	int jj2,minPos;


	jj2=tid;

	//global memory: organized as x_1,x_2,x_3,....,y_1,y_2,...,z_1,... to have coalescent access
	queryAux[0]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[1]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[2]=queryCUDA[jj2];

	int refIdx;
	for(int jj=0;jj<KNN;jj++) minDist[jj]=1e32;//equivalent to infinity. Thus, we know this element has not been assigned
	minDistThr=1e32;
	minPos=0;
	for(int ii=0;ii<ref_nb;ii++)
	{
		//__syncthreads();//to access constant memory coherently (this was effective in CUDA 3.2)
		refIdx = ii;
		

		distAux=(queryAux[0]-anchorCUDA[refIdx])*knn_v2_scaleCUDA[0];
		dist=distAux*distAux;
		refIdx += ref_nb;
		if(dist>minDistThr) continue;
		distAux=(queryAux[1]-anchorCUDA[refIdx])*knn_v2_scaleCUDA[1];
		dist+=distAux*distAux;
		refIdx += ref_nb;
		if(dist>minDistThr) continue;
		distAux=(queryAux[2]-anchorCUDA[refIdx])*knn_v2_scaleCUDA[2];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;


		//insert element" minimize memory exchanges
		minDist[minPos]=dist;
		indAux[minPos]=ii;
		findMaxPosition(minDist,&minDistThr,&minPos, KNN);
	}

	
	__syncthreads();//I need this to have coalescent memory access to inCUDA: speeds up the code by x4

	//copy indexes to global memory
	jj2=tid;
	for(int jj=0;jj<KNN;jj++)
	{
		//indCUDA[jj+jj2]=indAux[jj];
		indCUDA[jj2]=indAux[jj];
		jj2+=query_nb;
	}
	//copy distance if requested by user
	if(distCUDA != NULL)
	{
		jj2=tid;
		for(int jj=0;jj<KNN;jj++)
		{
			//indCUDA[jj+jj2]=indAux[jj];
			distCUDA[jj2]=minDist[jj];
			jj2+=query_nb;
		}
	}

}


//=============================================================================================================
int knnCUDA_v2(int *ind,float* dist, float *query,float *ref,long long int query_nb,int ref_nb, int KNN, float* scale, int devCUDA)
{
	// Variables and parameters
	//float* ref;                 // Pointer to reference point array: order is cache friednly with the GPU
	//float* query;               // Pointer to query point array: order is x1,y1,z1,x2,y2,z2... to be cache friendly
	//int*   ind;                 // Pointer to index array: size query_nb*knn_v2_maxKNN. Again, order is GPU cache friendly.
	//float*   dist;              // Pointer to distance^2 array: size query_nb*knn_v2_maxKNN. Again, order is GPU cache friendly. If pointer is null, scaled euclidean distance to each nearest neighbor is not returned
	//int    ref_nb       // Reference point number
	//int    query_nb    // Query point number
	//float scale[dimsImage] //
	
	if(dimsImage!=3)
	{
		printf("ERROR: at knnCUDA: code is not ready for dimsImage other than 3\n");//TODO: change this to any dimensionality
		return 2;
	}
	
	if(ref_nb <= 0)//nothing to do. There are no possible assignments
	{
		if(dist != NULL)
		{
			for(long long int ii = 0; ii < query_nb*KNN; ii++)
				dist[ii] = 1e32f;//no assignments
		}
		return 0;
	}
	//CUDA variables
	int *indCUDA;
	float* queryCUDA;
	float *anchorCUDA;
	float *distCUDA = NULL;
	
	//set CUDA device
	HANDLE_ERROR( hipSetDevice( devCUDA ) );
	
	
	// allocate memory on the GPU for the output: it will only be done once in the whole program
	HANDLE_ERROR( hipMalloc( (void**)&indCUDA, query_nb*KNN*sizeof(int) ) );//should it be a texture memory?NO. It does not fit in Cuda2Darray but it fits in linear 1Dtexture, although it does not seems to bring benefits
	HANDLE_ERROR( hipMalloc( (void**)&queryCUDA, query_nb*dimsImage*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&anchorCUDA, ref_nb*dimsImage*sizeof(float) ) );
	if( dist != NULL)
		HANDLE_ERROR( hipMalloc( (void**)&distCUDA, query_nb*KNN*sizeof(float) ) );

	// Copy image data to array
	HANDLE_ERROR(hipMemcpy(queryCUDA,query, dimsImage*query_nb*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(anchorCUDA,ref, dimsImage*ref_nb*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v2_scaleCUDA),scale, dimsImage * sizeof(float)));//constant memory	



	//prepare to launch kernel
	int numThreads=min(MAX_THREADS,(int) query_nb);
	int numGrids=min(MAX_BLOCKS,(int) (query_nb+numThreads-1)/numThreads);//TODO: play with these numbers to optimize

	knnKernelNoConstantMemory<<<numGrids,numThreads>>>(indCUDA, distCUDA, queryCUDA,anchorCUDA,ref_nb,query_nb, KNN);HANDLE_ERROR_KERNEL;	
	

	//copy results back
	HANDLE_ERROR(hipMemcpy(ind,indCUDA,query_nb*KNN*sizeof(int),hipMemcpyDeviceToHost));//retrieve indexes: memcopy is synchronous unless stated otherwise
	if( distCUDA != NULL)
		HANDLE_ERROR(hipMemcpy(dist,distCUDA,query_nb*KNN*sizeof(float),hipMemcpyDeviceToHost));


	//free memory
	HANDLE_ERROR( hipFree( indCUDA ) );
	HANDLE_ERROR( hipFree( queryCUDA ) );
	HANDLE_ERROR( hipFree( anchorCUDA ) );
	if( distCUDA != NULL)
		HANDLE_ERROR( hipFree( distCUDA ) );
	return 0;
}

#if 0
//===================================================================================================
int allocateGPUMemoryForKnnCUDA_(float *queryTemp,float **queryCUDA,int **indCUDA,long long int query_nb,float *scale, int KNN)
{
	
	
	HANDLE_ERROR( hipMalloc( (void**)&(*indCUDA), query_nb*KNN*sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(*queryCUDA), query_nb*dimsImage*sizeof(float) ) );
	// Copy image data to array
	HANDLE_ERROR(hipMemcpy((*queryCUDA),queryTemp, dimsImage*query_nb*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v2_scaleCUDA),scale, dimsImage * sizeof(float)));//constant memory
	return 0;
}

void setDeviceCUDA_(int devCUDA)
{
//WE ASSUME qeuryCUDA AND indCUDA HAVE BEEN ALLOCATED ALREADY AND MEMORY TRANSFERRED TO THE GPU
	HANDLE_ERROR( hipSetDevice( devCUDA ) );
}




//====================================================================================================
void deallocateGPUMemoryForKnnCUDA_(float **queryCUDA,int **indCUDA)
{
	HANDLE_ERROR( hipFree( *indCUDA ) );
	(*indCUDA)=NULL;
    HANDLE_ERROR( hipFree( *queryCUDA ) );
    (*queryCUDA)=NULL;
}
//==============================================================
void uploadknn_v2_ScaleCUDA_(float *scale)
{
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v2_scaleCUDA),scale, dimsImage * sizeof(float)));//constant memory
}

#endif