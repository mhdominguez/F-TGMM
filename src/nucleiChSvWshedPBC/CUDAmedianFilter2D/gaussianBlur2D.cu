#include "hip/hip_runtime.h"
/*
* Copyright (C) 2013 by  Fernando Amat
* See license.txt for full license and copyright notice.
*
* Authors: Fernando Amat 
*  gaussianBlur2D.cu
*
*  Created on: January 17th, 2013
*      Author: Fernando Amat
*
* \brief Code to calculate 2D median filter in CUDA using templates and different window sizes
*
*/

#include "gaussianBlur2D.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

//#define PROFILE_CODE // uncomment this to time execution

__constant__ int imDimCUDA[dimsImageSlice];//image dimensions



//=====================================================================================
template<class imgType, int radius>
__global__ void __launch_bounds__(BLOCK_SIDE*BLOCK_SIDE) gaussianBlurCUDAkernel(imgType* imCUDAin, imgType* imCUDAout, unsigned int imSize, float *kernel)
{

	const int radiusSize = ( 1 + 2 * radius) * ( 1 + 2 * radius);
	//shared memory to copy global memory
	__shared__ imgType blockNeigh [BLOCK_SIDE * BLOCK_SIDE];//stores values for a whole block
	imgType imgNeigh[ radiusSize ];//store values for each thread. This is the reason why radius is a template parameter and not a function input variable. Here we have a chance that everything fits in register memory for small radiuses
	
	
	int offset_x = blockIdx.x * (BLOCK_SIDE - 2* radius) - radius + threadIdx.x;//upper left corner of the image to start loading into share memory (counting overlap to accomodate radius)
	int offset_y = blockIdx.y * (BLOCK_SIDE - 2* radius) - radius + threadIdx.y;//upper left corner of the image to start loading into share memory (counting overlap to accomodate radius)
	
	int tid = threadIdx.y * BLOCK_SIDE + threadIdx.x;

	//each thread loads one pixel into share memory (colescent access)
	int pos;
	if( offset_x < 0 || offset_y < 0 || offset_x >= imDimCUDA[0] || offset_y >= imDimCUDA[1] )//out of bounds
	{
		pos = -1;
		blockNeigh[ tid ] = 0;//for now we assume zeros outside image boundaries
	}else{
		pos = offset_x + offset_y  * imDimCUDA[0];
		blockNeigh[ tid ] = imCUDAin[pos];
	}

	__syncthreads();

	if( threadIdx.x < radius || threadIdx.x >= BLOCK_SIDE-radius || threadIdx.y < radius || threadIdx.y >= BLOCK_SIDE-radius)
		return;//these threads are not needed (kind of a waste, but it is OK);

	
	//operate on block: this part could be substituted by any other operation in a blokc if we want to apply a different filter than median		
	int pp, count = 0;
	for( int ii = -radius; ii <= radius; ii++)
	{
		pp = threadIdx.x -radius + BLOCK_SIDE * ( threadIdx.y + ii );//initial position for jj for loop		

		for( int jj = -radius; jj <= radius; jj++)
		{				
			imgNeigh[count++] = blockNeigh[pp++];
		}
	}

	//selection algorithm to find the k-th smallest number (k = (radiusSize - 1) /2 (http://en.wikipedia.org/wiki/Selection_algorithm)
	imgType temp;
	for ( int ii=0; ii < (1 + radiusSize) /2; ii++)
	{
		// Find position of minimum element
		pp = ii;//minIndex
		for ( int jj = ii+1; jj < radiusSize; jj++)
		{
			if (imgNeigh[jj] < imgNeigh[pp])
			{
				pp = jj;
			}
		}
		temp = imgNeigh[pp];
		imgNeigh[pp] = imgNeigh[ii];
		imgNeigh[ii] = temp;
	}	

	if( pos>=0 && pos< imSize )
		imCUDAout[ pos ] = imgNeigh[ (radiusSize - 1) /2 ];
};



//===========================================================================

template<class imgType>
int gaussianBlurCUDA(imgType* im,int* imDim,int radius,int devCUDA)
{
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

	if( radius > (int)(floor( (BLOCK_SIDE -1) / 2.0f) )  || 2 * radius >= BLOCK_SIDE)
	{
		std::cout<<"ERROR: at gaussianBlurCUDA: code is not ready for such a large radius. Maximum radius allowed is "<<(int)(floor( (BLOCK_SIDE - 1) / 2.0f) )<<std::endl;
		return 2;
	}


	imgType* imCUDAinput = NULL;
	imgType* imCUDAoutput = NULL;


	int imSize = imDim[0];
	for( int ii = 1; ii < dimsImageSlice; ii++)
		imSize *= imDim[ii];

	//allocate memory in CUDA (input and output)
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAinput), imSize * sizeof(imgType) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAoutput), imSize * sizeof(imgType) ) );

	//transfer input: image and image dimensions
	HANDLE_ERROR(hipMemcpy(imCUDAinput, im, imSize * sizeof(imgType), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(imDimCUDA),imDim, dimsImageSlice * sizeof(int)));//constant memory

	//run kernel	
	dim3 threads( BLOCK_SIDE, BLOCK_SIDE );
	int numBlocks[dimsImageSlice];
	for (int ii = 0 ; ii< dimsImageSlice; ii++)
		numBlocks[ii] = (int) (ceil( (float)(imDim[ii] + radius ) / (float)(BLOCK_SIDE - 2 * radius) ) );
	dim3 blocks(numBlocks[0], numBlocks[1]);//enough to cover all the image

	switch(radius)
	{
	case 0:
		//do nothing
		break;
	case 1:
		gaussianBlurCUDAkernel<imgType, 1> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 2:
		gaussianBlurCUDAkernel<imgType, 2> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 3:
		gaussianBlurCUDAkernel<imgType, 3> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 4:
		gaussianBlurCUDAkernel<imgType, 4> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 5:
		gaussianBlurCUDAkernel<imgType, 5> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 6:
		gaussianBlurCUDAkernel<imgType, 6> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 7:
		gaussianBlurCUDAkernel<imgType, 7> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 8:
		gaussianBlurCUDAkernel<imgType, 8> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 9:
		gaussianBlurCUDAkernel<imgType, 9> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 10:
		gaussianBlurCUDAkernel<imgType, 10> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	case 11:
		gaussianBlurCUDAkernel<imgType, 11> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
		break;
	default:
		std::cout<<"ERROR: at gaussianBlurCUDA: code is not ready for such a large radius." <<std::endl;//If I need it at any point, I could extend this up to (int)(floor( (BLOCK_SIDE -1) / 2.0f) )
		return 4;
	}
	//copy result to host
	HANDLE_ERROR(hipMemcpy(im, imCUDAoutput, imSize * sizeof(imgType), hipMemcpyDeviceToHost));

	//deallocate memory
	HANDLE_ERROR( hipFree( imCUDAinput ) );
	HANDLE_ERROR( hipFree( imCUDAoutput ) );

	return 0;
}

//declare all the possible types so template compiles properly
template int gaussianBlurCUDA<unsigned char>(unsigned char* im,int* imDim,int radius,int devCUDA);
template int gaussianBlurCUDA<unsigned short int>(unsigned short int* im,int* imDim,int radius,int devCUDA);
template int gaussianBlurCUDA<float>(float* im,int* imDim,int radius,int devCUDA);


//===========================================================================

template<class imgType>
int gaussianBlurCUDASliceBySlice(imgType* im,int* imDim,float sigma,int devCUDA)
{
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

#ifdef PROFILE_CODE
	hipEvent_t start, stop;
	HANDLE_ERROR( hipEventCreate(&start ) );
	HANDLE_ERROR( hipEventCreate(&stop ) );
	HANDLE_ERROR( hipEventRecord(start,0 ) );
#endif

	int radius = getKernelRadiusForSigmaCUDA(sigma);
	
	//fill kernel
	float kernel[kradius*2+1];
	gaussianBlurKernel(sigma, kradius*2+1, kernel);	


	imgType* imCUDAinput = NULL;
	imgType* imCUDAoutput = NULL;


	int imSize = imDim[0];
	for( int ii = 1; ii < dimsImageSlice; ii++)
		imSize *= imDim[ii];

	//allocate memory in CUDA (input and output)
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAinput), imSize * sizeof(imgType) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAoutput), imSize * sizeof(imgType) ) );

	//kernel parameters
	dim3 threads( BLOCK_SIDE, BLOCK_SIDE );
	int numBlocks[dimsImageSlice];
	for (int ii = 0 ; ii< dimsImageSlice; ii++)
		numBlocks[ii] = (int) (ceil( (float)(imDim[ii] + radius ) / (float)(BLOCK_SIDE - 2*radius) ) );
	dim3 blocks(numBlocks[0], numBlocks[1]);//enough to cover all the image

	//perform median filter slice by slice
	for( int slice = 0; slice < imDim[dimsImageSlice ]; slice++)
	{

		//transfer input: image and image dimensions
		HANDLE_ERROR(hipMemcpy(imCUDAinput, im, imSize * sizeof(imgType), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(imDimCUDA),imDim, dimsImageSlice * sizeof(int)));//constant memory

		//run kernel			
		switch(radius)
		{
		case 0:
			//do nothing
			break;
		case 1:
			gaussianBlurCUDAkernel<imgType, 1> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		case 3:
			gaussianBlurCUDAkernel<imgType, 3> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		case 7:
			gaussianBlurCUDAkernel<imgType, 7> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		case 15:
			gaussianBlurCUDAkernel<imgType, 15> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		case 31:
			gaussianBlurCUDAkernel<imgType, 31> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		case 63:
			gaussianBlurCUDAkernel<imgType, 63> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		case 127:
			gaussianBlurCUDAkernel<imgType, 127> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, imSize);HANDLE_ERROR_KERNEL;
			break;
		default:
			std::cout<<"ERROR: at gaussianBlurCUDA: code is not ready for discrete radius "<<radius <<std::endl;//If I need it at any point, I could extend this up to (int)(floor( (BLOCK_SIDE -1) / 2.0f) )
			return 4;
		}
		//copy result to host
		HANDLE_ERROR(hipMemcpy(im, imCUDAoutput, imSize * sizeof(imgType), hipMemcpyDeviceToHost));

		im += imSize;//increment pointer to next slice
	}


	//deallocate memory
	HANDLE_ERROR( hipFree( imCUDAinput ) );
	HANDLE_ERROR( hipFree( imCUDAoutput ) );


#ifdef PROFILE_CODE	
	HANDLE_ERROR( hipEventRecord(stop,0 ) );
	HANDLE_ERROR( hipEventSynchronize(stop ) );

	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, start,stop ) );
	printf ("Time for the CUDA function gaussianBlurCUDASliceBySlice: %f ms\n", elapsedTime);
	HANDLE_ERROR( hipEventDestroy(start ) );
	HANDLE_ERROR( hipEventDestroy(stop ) );
#endif
	return 0;
}

//declare all the possible types so template compiles properly
template int gaussianBlurCUDASliceBySlice<unsigned char>(unsigned char* im,int* imDim,int radius,int devCUDA);
template int gaussianBlurCUDASliceBySlice<unsigned short int>(unsigned short int* im,int* imDim,int radius,int devCUDA);
template int gaussianBlurCUDASliceBySlice<float>(float* im,int* imDim,int radius,int devCUDA);

int getKernelRadiusForSigmaCUDA(float sigma) {
	int size = int(ceilf(sigma * 3)); //3 sigmas plus/minus should be decent
	if (size <= 1) {
		return 1;
	} else if ( size <= 3 ) {
		return 3;  
	} else if ( size <= 7 ) {
		return 7;
	} else if ( size <= 15 ) {
		return 15;
	} else if ( size <= 32 ) {
		return 31;
	} else if ( size <= 63 ) {
		return 63;		
	} else { //255 is max Kernel size
		return 127;
	}
}

void gaussianBlurKernel(float sigma, int size, float* kernel)
{
	float sigma2 = sigma * sigma;
	int middle = size / 2;
	for (int i = 0; i < size; ++i)
	{
		float distance = float (middle - i);
		float distance2 = distance * distance;
		float s = 1.0f / (sigma * SQRT_2PI * expf(-distance2 / (2.0f * sigma2));
		kernel[i] = s;
	}
}
