#include "hip/hip_runtime.h"
/*
* Copyright (C) 2013 by  Fernando Amat
* See license.txt for full license and copyright notice.
*
* Authors: Fernando Amat 
*  gaussianBlur2D.cu
*
*  Created on: January 17th, 2013
*      Author: Fernando Amat
*
* \brief Code to calculate 2D median filter in CUDA using templates and different window sizes
*
*/

#include "gaussianBlur2D.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>

//#define PROFILE_CODE // uncomment this to time execution

__constant__ int imDimCUDA[dimsImageSlice];//image dimensions

template<int radius>
__constant__ float kernelCUDA[ radius*2+1 ];

// how many threads per block in x (total num threads: x*y)
#define	ROWS_BLOCKDIM_X 16
// how many threads per block in y
#define	ROWS_BLOCKDIM_Y 16
// how many pixels in x are convolved by each thread
#define	ROWS_RESULT_STEPS 8
// these are the border pixels (loaded to support the kernel width for processing)
// the effective border width is ROWS_HALO_STEPS * ROWS_BLOCKDIM_X, which has to be
// larger or equal to the kernel radius to work
#define	ROWS_HALO_STEPS 1

#define	COLUMNS_BLOCKDIM_X 16
#define	COLUMNS_BLOCKDIM_Y 16
#define	COLUMNS_RESULT_STEPS 8
#define	COLUMNS_HALO_STEPS 1

#define	DEPTH_BLOCKDIM_X 16
#define	DEPTH_BLOCKDIM_Z 16
#define	DEPTH_RESULT_STEPS 8
#define	DEPTH_HALO_STEPS 1

static const float SQRT_2PI = sqrt(2*M_PI);


//=====================================================================================
template<class imgType, int radius>
__global__ void __launch_bounds__(MAX_THREADS_CUDA) gaussianBlurCUDAkernel(imgType* imCUDAin, imgType* imCUDAout, float *kernel, unsigned int imSize, bool convolve_y )
{
	//shared memory to copy global memory
	__shared__ double blockNeigh [MAX_THREADS_CUDA];//stores values for a whole block

	double sum;
	int offset_x;
	int offset_y;
	int tid;
	
	if (convolve_y) //y-dimension convolution
	{
		offset_y = blockIdx.y * (MAX_THREADS_CUDA - (2* radius+1)) -radius + threadIdx.x;
		offset_x = blockIdx.x;
		tid = threadIdx.x;
	}
	else //x-dimension convolution
	{
		offset_x = blockIdx.x * (MAX_THREADS_CUDA - (2* radius+1)) -radius + threadIdx.x;
		offset_y = blockIdx.y;
		tid = threadIdx.x;
	}

	//each thread loads one pixel into share memory (colescent access)
	int pos;
	if( offset_x < 0 || offset_y < 0 || offset_x >= imDimCUDA[0] || offset_y >= imDimCUDA[1] )//out of bounds
	{
		pos = -1;
		blockNeigh[ tid ] = 0;//zeros outside image boundaries
	}else{
		pos = offset_x + offset_y  * imDimCUDA[0];
		blockNeigh[ tid ] = (double)imCUDAin[pos];
	}
	__syncthreads();
	
	if( tid < radius || tid >= MAX_THREADS_CUDA-radius )
		return;//these threads are not needed (kind of a waste, but it is OK);	

	//here, we actually calculate the one-dimension convolution for this pixel
	int d;
	for( int k = -radius; k <= radius; k++)
	{
		d = tid + k;
		if(d >= 0 && d < MAX_THREADS_CUDA)
		{
			sum += (double)(blockNeigh[d] * kernelCUDA<radius>[radius - k]);
			//if (blockNeigh[d]>0)
			//	printf("found non-zero blockNeigh value at %d for {%d,%d}: %d\n",d,offset_x, offset_y, (int)blockNeigh[d] );
		}
	}

	if( pos>=0 && pos< imSize )
		imCUDAout[ pos ] = (imgType)(sum);
		//printf("%d: {%d, %d}/%d: %d/%d.\n",tid, offset_x, offset_y, pos, (int)imCUDAout[ pos ], (int)blockNeigh[tid]);
	
};



//===========================================================================

template<class imgType>
int gaussianBlurCUDA(imgType* im,int* imDim,float sigma,int devCUDA)
{
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

	int kradius = getKernelRadiusForSigmaCUDA(sigma);
	
	//fill kernel
	float kernel[kradius*2+1];
	gaussianBlurKernel(sigma, kradius*2+1, kernel);	

	imgType* imCUDAinput = NULL;
	imgType* imCUDAoutput = NULL;


	int imSize = imDim[0];
	for( int ii = 1; ii < dimsImageSlice; ii++)
		imSize *= imDim[ii];

	//allocate memory in CUDA (input and output)
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAinput), imSize * sizeof(imgType) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAoutput), imSize * sizeof(imgType) ) );

	//transfer input: image and image dimensions
	HANDLE_ERROR(hipMemcpy(imCUDAinput, im, imSize * sizeof(imgType), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(imDimCUDA),imDim, dimsImageSlice * sizeof(int)));//constant memory
	
	//run kernel	
	dim3 threads( BLOCK_SIDE, BLOCK_SIDE );
	int numBlocks[dimsImageSlice];
	for (int ii = 0 ; ii< dimsImageSlice; ii++)
		numBlocks[ii] = (int) (ceil( (float)(imDim[ii] + kradius ) / (float)(BLOCK_SIDE - 2 * kradius) ) );
	dim3 blocks(numBlocks[0], numBlocks[1]);//enough to cover all the image

	switch(kradius)
	{
	case 0:
		//do nothing
		break;
		case 1:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<1>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 1> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false );HANDLE_ERROR_KERNEL;
			break;
		case 3:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<3>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 3> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			break;
		case 7:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<7>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 7> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			break;
		case 15:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<15>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 15> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			break;
		case 31:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<31>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 31> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			break;
		case 63:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<63>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 63> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			break;
		case 127:
			hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<127>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 127> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			break;
	default:
		std::cout<<"ERROR: at gaussianBlurCUDA: code is not ready for discrete radius "<<kradius <<std::endl;//If I need it at any point, I could extend this up to (int)(floor( (BLOCK_SIDE -1) / 2.0f) )
		return 4;
	}
	//copy result to host
	HANDLE_ERROR(hipMemcpy(im, imCUDAoutput, imSize * sizeof(imgType), hipMemcpyDeviceToHost));

	//deallocate memory
	HANDLE_ERROR( hipFree( imCUDAinput ) );
	HANDLE_ERROR( hipFree( imCUDAoutput ) );

	return 0;
}

//declare all the possible types so template compiles properly
template int gaussianBlurCUDA<unsigned char>(unsigned char* im,int* imDim,float sigma,int devCUDA);
template int gaussianBlurCUDA<unsigned short int>(unsigned short int* im,int* imDim,float sigma,int devCUDA);
template int gaussianBlurCUDA<float>(float* im,int* imDim,float sigma,int devCUDA);


//===========================================================================

template<class imgType>
int gaussianBlurCUDASliceBySlice(imgType* im,int* imDim,float sigma,int devCUDA)
{
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

#ifdef PROFILE_CODE
	hipEvent_t start, stop;
	HANDLE_ERROR( hipEventCreate(&start ) );
	HANDLE_ERROR( hipEventCreate(&stop ) );
	HANDLE_ERROR( hipEventRecord(start,0 ) );
#endif

	int kradius = getKernelRadiusForSigmaCUDA(sigma);
	
	//fill kernel
	float kernel[kradius*2+1];
	gaussianBlurKernel(sigma, kradius*2+1, kernel);	


	imgType* imCUDAinput = NULL;
	imgType* imCUDAoutput = NULL;


	int imSize = imDim[0];
	for( int ii = 1; ii < dimsImageSlice; ii++)
		imSize *= imDim[ii];

	//allocate memory in CUDA (input and output)
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAinput), imSize * sizeof(imgType) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(imCUDAoutput), imSize * sizeof(imgType) ) );

	//copy imDim as constant
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(imDimCUDA),imDim, dimsImageSlice * sizeof(int)));//constant memory	
	
	dim3 threads( MAX_THREADS_CUDA ); //we will go line-by-line or column-by-column, or shorter than that, as dictated by max threads
	dim3 blocks;

	
	//perform separable convolution slice by slice
	for( int slice = 0; slice < imDim[dimsImageSlice ]; slice++)
	{
		//transfer input: image and image dimensions
		HANDLE_ERROR(hipMemcpy(imCUDAinput, im, imSize * sizeof(imgType), hipMemcpyHostToDevice));
		
		//run kernel			
		switch(kradius)
		{
		case 0:
			//do nothing
			break;
		case 1:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];
			if ( slice == 0 )
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<1>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 1> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 1> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		case 3:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];
			if ( slice == 0 )
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<3>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 3> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 3> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		case 7:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];
			if ( slice == 0 )
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<7>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 7> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 7> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		case 15:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];	
			if ( slice == 0 )
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<15>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 15> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 15> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		case 31:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];	
			if ( slice == 0 )
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<31>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 31> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 31> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		case 63:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];			
			if ( slice == 0 )
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<63>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 63> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 63> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		case 127:
			blocks.x = (int) (ceil( (float)(imDim[0]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			blocks.y = imDim[1];
			if ( slice == 0 )			
				hipMemcpyToSymbol(HIP_SYMBOL(  kernelCUDA<127>), kernel, (kradius*2+1) * sizeof(float) );
			gaussianBlurCUDAkernel<imgType, 127> <<<blocks, threads>>>(imCUDAinput, imCUDAoutput, kernel, imSize, false);HANDLE_ERROR_KERNEL;
			blocks.x = imDim[0];
			blocks.y = (int) (ceil( (float)(imDim[1]+kradius) / (float)(MAX_THREADS_CUDA - (2 * kradius+1)) ) );
			gaussianBlurCUDAkernel<imgType, 127> <<<blocks, threads>>>(imCUDAoutput, imCUDAinput, kernel, imSize, true);HANDLE_ERROR_KERNEL;
			break;
		default:
			std::cout<<"ERROR: at gaussianBlurCUDA: code is not ready for discrete radius "<<kradius <<std::endl;//If I need it at any point, I could extend this up to (int)(floor( (BLOCK_SIDE -1) / 2.0f) )
			return 4;
		}
		//copy result to host
		HANDLE_ERROR(hipMemcpy(im, imCUDAinput, imSize * sizeof(imgType), hipMemcpyDeviceToHost));
	
		im += imSize;//increment pointer to next slice
	}


	//deallocate memory
	HANDLE_ERROR( hipFree( imCUDAinput ) );
	HANDLE_ERROR( hipFree( imCUDAoutput ) );


#ifdef PROFILE_CODE	
	HANDLE_ERROR( hipEventRecord(stop,0 ) );
	HANDLE_ERROR( hipEventSynchronize(stop ) );

	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime(&elapsedTime, start,stop ) );
	printf ("Time for the CUDA function gaussianBlurCUDASliceBySlice: %f ms\n", elapsedTime);
	HANDLE_ERROR( hipEventDestroy(start ) );
	HANDLE_ERROR( hipEventDestroy(stop ) );
#endif
	return 0;
}

//declare all the possible types so template compiles properly
template int gaussianBlurCUDASliceBySlice<unsigned char>(unsigned char* im,int* imDim,float sigma,int devCUDA);
template int gaussianBlurCUDASliceBySlice<unsigned short int>(unsigned short int* im,int* imDim,float sigma,int devCUDA);
template int gaussianBlurCUDASliceBySlice<float>(float* im,int* imDim,float sigma,int devCUDA);

int getKernelRadiusForSigmaCUDA(float sigma) {
	int size = int(ceilf(sigma * 3)); //3 sigmas plus/minus should be decent
	if (size <= 1) {
		return 1;
	} else if ( size <= 3 ) {
		return 3;  
	} else if ( size <= 7 ) {
		return 7;
	} else if ( size <= 15 ) {
		return 15;
	} else if ( size <= 32 ) {
		return 31;
	} else if ( size <= 63 ) {
		return 63;		
	} else { //255 is max Kernel size
		return 127;
	}
}

void gaussianBlurKernel(float sigma, int size, float* kernel)
{
	float sigma2 = 2.0f * sigma * sigma;
	float sigma1 = 1.0f / sqrt( M_PI * sigma2 ); 
	int middle = size / 2;
	//std::cout << "GB kernel: ";
	for (int i = 0; i < size; ++i)
	{
		float distance = float (middle - i);
		//float distance2 = distance * distance;
		//float s = 1.0f / (sigma * sqrtf(2.0f * PI_F)) * expf(-distance2 / (2.0f * sigma2));
		kernel[i] = sigma1 * exp(-(distance * distance) / sigma2);
		//std::cout << i << ":" << kernel[i] << ",";
	}
	//std::cout << std::endl;
}
