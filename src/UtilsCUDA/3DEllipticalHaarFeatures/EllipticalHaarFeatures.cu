#include "hip/hip_runtime.h"
#include "EllipticalHaarFeatures.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
/* Includes for HealPix*/
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <vector>

using namespace std;

__device__ static const double PI_= 3.14159265358979311600;
__device__ static const double SQRT3_= 1.73205080756887719318;

__constant__  double r0CUDA[1];//initial radius (based on sigma) to define central cell ellipsoid
__constant__  double kSigmaCUDA[1];//factor to define outer rings dimensions in the C-HoG block (r_m=(r0+m*kSigma)*sigma)
__constant__  int numCellsHEALPixCUDA[1]; // number of sectors per fixed radius using HEALpix

__constant__ int x2pix[128];//necessary for HEALPIX routine
__constant__ int y2pix[128];

__device__ static const double piover2 = 1.5707963267948966; 
__device__ static const double twopi = 6.2831853071795862;
__device__ static const int    ns_max = 8192;
  

__constant__ float kernelSeparableConv[1+2*maxRadiusBox];//kernel for separable convolution. It does not make sense if it is larger than the box

texture<imageType, dimsImage, hipReadModeElementType> textureImage; 


static const float constDoG_0 = 2.0f*sqrt(2.0f); //some ocnstants we need for DoG kernel
static const float constDoG_1 =(4.0f*sqrt(2.0f*log(2.0f)));

int basicEllipticalHaarFeatureVector::numCells;
int basicEllipticalHaarFeatureVector::numRings;

static int countBoxesDebug = 0;//global counter for debugging purposes




//=================================================================================================
//===================================HealPix function executed in GPU=============================
/* -----------------------------------------------------------------------------
 *
 *  Copyright (C) 1997-2010 Krzysztof M. Gorski, Eric Hivon,
 *                          Benjamin D. Wandelt, Anthony J. Banday, 
 *                          Matthias Bartelmann, 
 *                          Reza Ansari & Kenneth M. Ganga 
 *
 *
 *  This file is part of HEALPix.
 *
 *  HEALPix is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  HEALPix is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with HEALPix; if not, write to the Free Software
 *  Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA  02110-1301  USA
 *
 *  For more information about HEALPix see http://healpix.jpl.nasa.gov
 *
 *----------------------------------------------------------------------------- */
/* vec2pix_nest.c */

/* Local Includes */
//#include "chealpix.h"
void mk_xy2pix(int *x2pix, int *y2pix) {
  /* =======================================================================
   * subroutine mk_xy2pix
   * =======================================================================
   * sets the array giving the number of the pixel lying in (x,y)
   * x and y are in {1,128}
   * the pixel number is in {0,128**2-1}
   *
   * if  i-1 = sum_p=0  b_p * 2^p
   * then ix = sum_p=0  b_p * 4^p
   * iy = 2*ix
   * ix + iy in {0, 128**2 -1}
   * =======================================================================
   */
  int i, K,IP,I,J,ID;
  
  for(i = 0; i < 127; i++) x2pix[i] = 0;
  for( I=1;I<=128;I++ ) {
    J  = I-1;//            !pixel numbers
    K  = 0;//
    IP = 1;//
    truc : if( J==0 ) {
      x2pix[I-1] = K;
      y2pix[I-1] = 2*K;
    }
    else {
      ID = (int)fmod((float)J,2.0f);
      J  = J/2;
      K  = IP*ID+K;
      IP = IP*4;
      goto truc;
    }
  }     
  
}
__device__ void vec2pix_nest( const long long int nside, double *vec, long long int *ipix) 
{

  /* =======================================================================
   * subroutine vec2pix_nest(nside, vec, ipix)
   * =======================================================================
   * gives the pixel number ipix (NESTED) corresponding to vector vec
   *
   * the computation is made to the highest resolution available (nside=8192)
   * and then degraded to that required (by integer division)
   * this doesn't cost more, and it makes sure that the treatement of round-off 
   * will be consistent for every resolution
   * =======================================================================
   */
  
  double z, za, z0, tt, tp, tmp, phi;
  int    face_num = 0,jp = 0,jm = 0;
  long long int   ifp = 0, ifm = 0;
  int    ix = 0, iy = 0, ix_low = 0, ix_hi = 0, iy_low = 0, iy_hi = 0, ipf = 0, ntt = 0;
  //double piover2 = 0.5*PI_, twopi = 2.0*PI_;
  //int    ns_max = 8192;
  //int x2pix[128], y2pix[128]; //set as __constant__ in the GPU since they are the same for all calls
  //static int x2pix[128], y2pix[128];
  //static char setup_done = 0;
  
  //if( nside<1 || nside>ns_max ) {
  //  fprintf(stderr, "%s (%d): nside out of range: %ld\n", __FILE__, __LINE__, nside);
  //  exit(0);
  //}

  //if( !setup_done ) {
    //mk_xy2pix(x2pix,y2pix);
    //setup_done = 1;
  //}
  
  z   = vec[2]/sqrt(vec[0]*vec[0] + vec[1]*vec[1] + vec[2]*vec[2]);
  phi = 0.0;
  if (vec[0] != 0.0 || vec[1] != 0.0) {
    phi   = atan2(vec[1],vec[0]); /* in ]-pi, pi] */
    if (phi < 0.0) phi += twopi; /* in  [0, 2pi[ */
  }

  za = fabs(z);
  z0 = 2./3.;
  tt = phi / piover2; /* in [0,4[ */
  
  if( za<=z0 ) { /* equatorial region */
    
    /* (the index of edge lines increase when the longitude=phi goes up) */
    jp = (int)floor(ns_max*(0.5 + tt - z*0.75)); /* ascending edge line index */
    jm = (int)floor(ns_max*(0.5 + tt + z*0.75)); /* descending edge line index */
    
    /* finds the face */
    ifp = jp / ns_max; /* in {0,4} */
    ifm = jm / ns_max;
    
    if( ifp==ifm ) face_num = (int)fmod((float)ifp,4.0f) + 4; /* faces 4 to 7 */
    else if( ifp<ifm ) face_num = (int)fmod((float)ifp,4.0f); /* (half-)faces 0 to 3 */
    else face_num = (int)fmod((float)ifm,4.0f) + 8;           /* (half-)faces 8 to 11 */
    
    ix = (int)fmod((float)jm, (float)ns_max);
    iy = ns_max - (int)fmod((float)jp, (float)ns_max) - 1;
  }
  else { /* polar region, za > 2/3 */
    
    ntt = (int)floor(tt);
    if( ntt>=4 ) ntt = 3;
    tp = tt - ntt;
    tmp = sqrt( 3.*(1. - za) ); /* in ]0,1] */
    
    /* (the index of edge lines increase when distance from the closest pole
     * goes up)
     */
    /* line going toward the pole as phi increases */
    jp = (int)floor( ns_max * tp          * tmp ); 

    /* that one goes away of the closest pole */
    jm = (int)floor( ns_max * (1. - tp) * tmp );
    jp = (int)(jp < ns_max-1 ? jp : ns_max-1);
    jm = (int)(jm < ns_max-1 ? jm : ns_max-1);
    
    /* finds the face and pixel's (x,y) */
    if( z>=0 ) {
      face_num = ntt; /* in {0,3} */
      ix = ns_max - jm - 1;
      iy = ns_max - jp - 1;
    }
    else {
      face_num = ntt + 8; /* in {8,11} */
      ix =  jp;
      iy =  jm;
    }
  }
  
  ix_low = (int)fmod((float)ix,128.0f);
  ix_hi  =     ix/128;
  iy_low = (int)fmod((float)iy,128.0f);
  iy_hi  =     iy/128;


  ipf = (x2pix[ix_hi]+y2pix[iy_hi]) * (128 * 128)+ (x2pix[ix_low]+y2pix[iy_low]);
  ipf = (long long int)(ipf / pow((float)ns_max/nside,2));     /* in {0, nside**2 - 1} */
  *ipix =(long long int)( ipf + face_num*pow((float)nside,2)); /* in {0, 12*nside**2 - 1} */
  
}



//===============================================================================================


//=======================================================================================================
//===========================================================================
//eigen value functions: I need to have everythign in one file

//=============================================================================================
//=========================================================================
//determinant for 3x3 symmetric matrix
__device__  double determinantSymmetricW_3D(const double *W_k)
{
	return W_k[0]*(W_k[3]*W_k[5]-W_k[4]*W_k[4])-W_k[1]*(W_k[1]*W_k[5]-W_k[2]*W_k[4])+W_k[2]*(W_k[1]*W_k[4]-W_k[2]*W_k[3]);
}
//=========================================================================
//inverse for a 3x3 symmetric matrix
__device__  void inverseSymmetricW_3D(double *W,double *W_inverse)
{
	double detW=determinantSymmetricW_3D(W);
	if(fabs(detW)<1e-16) //matrix is singular
	{
		W_inverse[0]=1e300;W_inverse[1]=1e300;W_inverse[2]=1e300;W_inverse[3]=1e300;W_inverse[4]=1e300;W_inverse[5]=1e300;
		return;
	}
	W_inverse[0]=(W[3]*W[5]-W[4]*W[4])/detW;
	W_inverse[1]=(W[4]*W[2]-W[1]*W[5])/detW;
	W_inverse[2]=(W[1]*W[4]-W[3]*W[2])/detW;


	W_inverse[3]=(W[0]*W[5]-W[2]*W[2])/detW;
	W_inverse[4]=(W[1]*W[2]-W[0]*W[4])/detW;

	W_inverse[5]=(W[0]*W[3]-W[1]*W[1])/detW;

	return;
}
//analytical solution for eigenvalues 3x3 real symmetric matrices
//formula for eigenvalues from http://en.wikipedia.org/wiki/Eigenvalue_algorithm#Eigenvalues_of_3.C3.973_matrices
__device__  void  eig3(const double *w, double *d, double *v)
{
	double m,p,q;
	int vIsZero=0;
	double phi,aux1,aux2,aux3;



	
	if (w[1] * w[1] + w[2] * w[2] + w[4] * w[4] == 0.0) //check for special case when matrix ix diagonal
	{
		d[0] = w[0];
		d[1] = w[3];
		d[2] = w[5];
		v[0] = 1.0; v[1] = 0.0; v[2] = 0.0;
		v[3] = 0.0; v[4] = 1.0; v[5] = 0.0;
		v[6] = 0.0; v[7] = 0.0; v[8] = 1.0;
		return;
	}else if (fabs(determinantSymmetricW_3D(w))<1e-24)//calculate determinant to check if matrix is singular we consider matrix is singular
	{
		d[0]=0.0;
		//solve a quadratic equation
		m=-w[0]-w[3]-w[5];
		q=-w[1]*w[1]-w[2]*w[2]-w[4]*w[4]+w[0]*w[3]+w[0]*w[5]+w[3]*w[5];
		p=m*m-4.0*q;
		if(p<0) p=0.0;//to avoid numerical errors (symmetric matrix should have real eigenvalues)
		else p=sqrt(p);
		d[1]=0.5*(-m+p);
		d[2]=0.5*(-m-p);

	}else{//matrix not singular
		m=(w[0]+w[3]+w[5])/3.0;//trace of w /3
		q=0.5*((w[0]-m)*((w[3]-m)*(w[5]-m)-w[4]*w[4])-w[1]*(w[1]*(w[5]-m)-w[2]*w[4])+w[2]*(w[1]*w[4]-w[2]*(w[3]-m)));//determinant(a-mI)/2
		p=(2.0*(w[1]*w[1]+w[2]*w[2]+w[4]*w[4])+(w[0]-m)*(w[0]-m)+(w[3]-m)*(w[3]-m)+(w[5]-m)*(w[5]-m))/6.0;


		//NOTE: the follow formula assume accurate computation and therefor q/p^(3/2) should be in range of [1,-1],
		//but in real code, because of numerical errors, it must be checked. Thus, in case abs(q) >= abs(p^(3/2)), set phi = 0;
		phi= q / pow(p,1.5);
		if(phi <= -1)
			phi = PI_ / 3.0;
		else if (phi >= 1)
			phi = 0;
		else 
			phi = acos(phi)/3.0;

		aux1=cos(phi);aux2=sin(phi);aux3=sqrt(p);

		//eigenvalues
		d[0] = m + 2.0*aux3*aux1;
		d[1] = m - aux3*(aux1 + SQRT3_*aux2);
		d[2] = m - aux3*(aux1 - SQRT3_*aux2);
	}

	//eigenvectors
	v[0]=w[1]*w[4]-w[2]*(w[3]-d[0]);v[1]=w[2]*w[1]-w[4]*(w[0]-d[0]);v[2]=(w[0]-d[0])*(w[3]-d[0])-w[1]*w[1];
	v[3]=w[1]*w[4]-w[2]*(w[3]-d[1]);v[4]=w[2]*w[1]-w[4]*(w[0]-d[1]);v[5]=(w[0]-d[1])*(w[3]-d[1])-w[1]*w[1];
	v[6]=w[1]*w[4]-w[2]*(w[3]-d[2]);v[7]=w[2]*w[1]-w[4]*(w[0]-d[2]);v[8]=(w[0]-d[2])*(w[3]-d[2])-w[1]*w[1];

	//normalize eigenvectors
	phi=sqrt(v[0]*v[0]+v[1]*v[1]+v[2]*v[2]);
	if(phi>1e-12){ v[0]/=phi;v[1]/=phi;v[2]/=phi;}
	else{//numerically seems zero: we need to try the other pair of vectors to form the null space (it could be that v1 and v2 were parallel)
		v[0]=w[1]*(w[5]-d[0])-w[2]*w[4];v[1]=w[2]*w[2]-(w[5]-d[0])*(w[0]-d[0]);v[2]=(w[0]-d[0])*w[4]-w[1]*w[2];
		phi=sqrt(v[0]*v[0]+v[1]*v[1]+v[2]*v[2]);
		if(phi>1e-12){v[0]/=phi;v[1]/=phi;v[2]/=phi;}
		else vIsZero+=1;
	}    


	phi=sqrt(v[3]*v[3]+v[4]*v[4]+v[5]*v[5]);
	if(phi>1e-12){ v[3]/=phi;v[4]/=phi;v[5]/=phi;}
	else{//numerically seems zero: we need to try the 
		v[3]=w[1]*(w[5]-d[1])-w[2]*w[4];v[4]=w[2]*w[2]-(w[5]-d[1])*(w[0]-d[1]);v[5]=(w[0]-d[1])*w[4]-w[1]*w[2];
		phi=sqrt(v[3]*v[3]+v[4]*v[4]+v[5]*v[5]);
		if(phi>1e-12){v[3]/=phi;v[4]/=phi;v[5]/=phi;}
		else vIsZero+=2;
	}

	phi=sqrt(v[6]*v[6]+v[7]*v[7]+v[8]*v[8]);
	if(phi>1e-12) {v[6]/=phi;v[7]/=phi;v[8]/=phi;}
	else{//numerically seems zero: we need to try the 
		v[6]=w[1]*(w[5]-d[2])-w[2]*w[4];v[7]=w[2]*w[2]-(w[5]-d[2])*(w[0]-d[2]);v[8]=(w[0]-d[2])*w[4]-w[1]*w[2];
		phi=sqrt(v[6]*v[6]+v[7]*v[7]+v[8]*v[8]);
		if(phi>1e-12){v[6]/=phi;v[7]/=phi;v[8]/=phi;}
		else vIsZero+=4;
	}

	//adjust v in case zome eigenvalues are zeros
	switch(vIsZero)
	{
	case 1:
		v[0]=v[4]*v[8]-v[5]*v[7];
		v[1]=v[5]*v[6]-v[3]*v[8];
		v[2]=v[4]*v[6]-v[3]*v[7];
		break;

	case 2:
		v[3]=v[1]*v[8]-v[2]*v[7];
		v[4]=v[2]*v[6]-v[0]*v[8];
		v[5]=v[1]*v[6]-v[0]*v[7];
		break;

	case 4:
		v[6]=v[4]*v[2]-v[5]*v[1];
		v[7]=v[5]*v[0]-v[3]*v[2];
		v[8]=v[4]*v[0]-v[3]*v[1];
		break;
	case 3:
		phi=sqrt(v[7]*v[7]+v[6]*v[6]);
		if(phi<1e-12)//it means first eigenvector is [0 0 1]
		                                              {v[3]=1.0;v[4]=0.0;v[5]=0.0;}
		else{ v[3]=-v[7]/phi;v[4]=v[6]/phi;v[5]=0.0;}
		v[0]=v[4]*v[8]-v[5]*v[7];
		v[1]=v[5]*v[6]-v[3]*v[8];
		v[2]=v[3]*v[7]-v[4]*v[6];
		break;

	case 6:
		phi=sqrt(v[1]*v[1]+v[0]*v[0]);
		if(phi<1e-12)//it means first eigenvector is [0 0 1]
		{v[6]=1.0;v[7]=0.0;v[8]=0.0;}
		else{ v[6]=-v[1]/phi;v[7]=v[0]/phi;v[8]=0.0;}
		v[3]=v[1]*v[8]-v[2]*v[7];
		v[4]=v[2]*v[6]-v[0]*v[8];
		v[5]=v[0]*v[7]-v[1]*v[6];
		break;

	case 5:
		phi=sqrt(v[4]*v[4]+v[5]*v[5]);
		if(phi<1e-12)//it means first eigenvector is [0 0 1]
		{v[0]=1.0;v[1]=0.0;v[2]=0.0;}
		else{ v[0]=-v[4]/phi;v[1]=v[5]/phi;v[2]=0.0;}
		v[6]=v[4]*v[2]-v[5]*v[1];
		v[7]=v[5]*v[0]-v[3]*v[2];
		v[8]=v[1]*v[3]-v[4]*v[0];
		break;

	case 7://matrix is basically zero: so we set eigenvectors to identity matrix
		v[1]=v[2]=v[3]=v[5]=v[6]=v[7]=0.0;
		v[0]=v[4]=v[8]=1.0;
		break;

	}

	//make sure determinant is +1 for teh rotation matrix
	phi=v[0]*(v[4]*v[8]-v[5]*v[7])-v[1]*(v[3]*v[8]-v[5]*v[6])+v[2]*(v[3]*v[7]-v[4]*v[6]);
	if(phi<0)
	{
		v[0]=-v[0];v[1]=-v[1];v[2]=-v[2];
	}	
}
//----------------------------------------------------------------
//-------------------------------------------------------------------
//analytical solution for eigenvalues 2x2 real symmetric matrices
 __device__ void eig2(const double *w, double *d, double *v)
{
	double aux1,phi;
	int vIsZero=0;

	aux1=(w[0]+w[2])/2.0;
	phi=sqrt(4.0*w[1]*w[1] + (w[0]-w[2])*(w[0]-w[2]))/2.0;

	d[0] = aux1 + phi;
	d[1] = aux1 - phi;


	//calculate eigenvectors
	//eigenvectors
	v[0]=-w[1];v[1]=w[0]-d[0];
	v[2]=-w[1];v[3]=w[0]-d[1];

	//normalize eigenvectors
	phi=sqrt(v[0]*v[0]+v[1]*v[1]);
	if(phi>0){ v[0]/=phi;v[1]/=phi;}
	else vIsZero+=1;

	phi=sqrt(v[2]*v[2]+v[3]*v[3]);
	if(phi>0){ v[2]/=phi;v[3]/=phi;}
	else vIsZero+=2;

	switch(vIsZero)
	{
	case 1:
		v[0]=-v[3];v[1]=v[2];
		break;
	case 2:
		v[2]=-v[1];v[3]=v[0];
		break;
	case 3://matrix is basically zero: so we set eigenvectors to identity matrix
		v[1]=v[2]=0.0;
		v[0]=v[3]=1.0;
		break;
	}
	//make sure determinant is +1 for teh rotation matrix
	phi=v[0]*v[3]-v[1]*v[2];
	if(phi<0)
	{
		v[0]=-v[0];v[1]=-v[1];
	}
}


//===============================end of eigenvalus functionality============================================================

//===============================beginning of eigenvalues kernel============================================================
 //order indicates if we want to order the elements according to eigenvectors.
 //order=0 ->no need to order them; order<0 -> order in descend order;order>0->order in ascendant order
 __global__ void __launch_bounds__(MAX_THREADS_CUDA) computeEigenDecompositionKernel(double *W,int numEllipsoids,double *dCUDA,double* vCUDA, int order)
{
	double v[dimsImage*dimsImage];
	double d[dimsImage];
	double Wlocal[dimsImage*(dimsImage+1)/2];

	
	long long int tid=threadIdx.x + blockIdx.x * blockDim.x;
	long long int pos=tid;

	if(tid<numEllipsoids)
	{
		
		//copy from global memory
		pos=tid;
		for(int ii=0;ii<dimsImage*(dimsImage+1)/2;ii++)
		{
			Wlocal[ii] = W[pos];
			pos+=numEllipsoids;
		}
		if(dimsImage == 3)
			eig3(Wlocal, d, v);
		else if (dimsImage == 2)
			eig2(Wlocal, d, v);

		//order according to eigenvalues using bubble sort
		double dAux,vAux[dimsImage];
		int flag = 1;//indicates if any swapping has occurred
		int posAux;
		if(order<0)//descend order
		{
			while(flag == 1)
			{
				flag =0;
				for(int ii=0;ii<dimsImage-1;ii++)
				{
					if(d[ii]<d[ii+1])//swap
					{
						flag = 1;
						dAux = d[ii];d[ii] = d[ii+1]; d[ii+1] = dAux;//swap eigenvalues
						posAux = ii*dimsImage;
						for(int jj=0;jj<dimsImage;jj++)//swap eigenvectors
						{
							vAux[jj]=v[posAux];
							v[posAux] = v[posAux+dimsImage];
							v[posAux+dimsImage] = vAux[jj];
							posAux++;
						}
					}
				}
			}
		}else if(order>0)//ascendant order
		{
			while(flag == 1)
			{
				flag =0;
				for(int ii=0;ii<dimsImage-1;ii++)
				{
					if(d[ii]>d[ii+1])//swap
					{
						flag = 1;
						dAux = d[ii];d[ii] = d[ii+1]; d[ii+1] = dAux;//swap eigenvalues
						posAux = ii*dimsImage;
						for(int jj=0;jj<dimsImage;jj++)//swap eigenvectors
						{
							vAux[jj]=v[posAux];
							v[posAux] = v[posAux+dimsImage];
							v[posAux+dimsImage] = vAux[jj];
							posAux++;
						}
					}
				}
			}
		}

		__syncthreads();//to ensure coalescent memory access
		//copy back to global memory
		pos=tid;
		for(int ii=0;ii<dimsImage;ii++)
		{
			dCUDA[pos] = d[ii];//for coalescencent access efficiency
			pos+=numEllipsoids;
		}
		pos=tid;
		for(int ii=0;ii<dimsImage*dimsImage;ii++)
		{
			vCUDA[pos] = v[ii];
			pos+=numEllipsoids;
		}
	}
}

 //symmetrize (flip direction) if necessary
 __global__ void __launch_bounds__(MAX_THREADS_CUDA) applySymmetryToEigenvectorsKernel(double* vCUDA,int numEllipsoids,int symmetry)
 {
	 int tid=threadIdx.x + blockIdx.x * blockDim.x;
	 int vCUDAsize = numEllipsoids*dimsImage*dimsImage; 

	 if(tid<vCUDAsize)
	 {
		 //find out in which position we are within vCUDA we are in
		 int ellipsoidIdx = tid%numEllipsoids;
		 int vPos = (tid-ellipsoidIdx)/numEllipsoids;

		 //decide if it needs to be flipped or not
		 switch(vPos%dimsImage)
		 {
		 case 0://this is a v element that will multiply X value so we need to check if X needs to be flipped
			 if((symmetry & 0x01))
				 vCUDA[tid] *= -1.0;
			 break;
		 case 1://this is a v element that will multiply Y value so we need to check if Y needs to be flipped
			 if((symmetry & 0x02))
				 vCUDA[tid] *= -1.0;
			 break;
		 case 2://this is a v element that will multiply Z value so we need to check if Z needs to be flipped
			 if((symmetry & 0x04))
				 vCUDA[tid] *= -1.0;
			 break;
		 }		 
	 }
 }
//===============================end of eigenvalues kernel============================================================

//==============================beginning of interpolation kernel===============================================
//when you defined the properties of the texture textureImage you decide what to do with out of bounds values and if you prefer linear or nearest neighbor interpolation
__global__ void __launch_bounds__(MAX_THREADS_CUDA) interpolate3DBoxKernel(double* mCUDA,double* vCUDA,int ellipseIdx,int numEllipsoids,float* boxCUDA,int radiusBox_0, int radiusBox_1, int radiusBox_2,float* meanBoxCUDA,float* stdBoxCUDA)
{
	int boxSize=(1+2*radiusBox_0)*(1+2*radiusBox_1)*(1+2*radiusBox_2);
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	__shared__ float m[dimsImage];
	__shared__ float v[dimsImage*dimsImage];//eigen vectors of the covariance matrix

	__shared__ float meanBox[MAX_THREADS_CUDA];//stores partial sums of the intensity to normalize the box
	__shared__ float stdBox[MAX_THREADS_CUDA];//stores partial sums of the intensity^2 to normalize the box

	//read from global memory to shared memory: unfortunately we do not have coalescent access
	if(threadIdx.x < dimsImage)
	{
		m[threadIdx.x]=mCUDA[threadIdx.x*numEllipsoids+ellipseIdx];
	}else if(threadIdx.x <dimsImage*(1+dimsImage))
	{
		v[threadIdx.x-dimsImage] = vCUDA[(threadIdx.x-dimsImage)*numEllipsoids+ellipseIdx];
	}

	meanBox[threadIdx.x]=0.0f;
	stdBox[threadIdx.x]=0.0f;
	__syncthreads();

	//calculate interpolation
	if(tid<boxSize)
	{
		float val,xi,yi,zi;
		int x,y,z,aux,aux2;

		aux=(1+2*radiusBox_0);
		x=tid%aux;
		aux2=(tid-x)/aux;
		aux=(1+2*radiusBox_1);
		y=aux2%aux;


		//if(dimsImage==3)
		//{
			z=(aux2-y)/aux;
			//center coordinates
			x-=radiusBox_0;
			y-=radiusBox_1;
			z-=radiusBox_2;
			xi=v[0]*x+v[1]*y+v[2]*z+m[0];
			yi=v[3]*x+v[4]*y+v[5]*z+m[1];
			zi=v[6]*x+v[7]*y+v[8]*z+m[2];
			val=tex3D(textureImage,xi+0.5f,yi+0.5f,zi+0.5f);//For CUDA, NN means just ceiling the coordinates, so we add 0.5 to really make it NN

		//}
		/*
		else if(dimsImage==2)
		{
			xi=v[0]*x+v[1]*y+m[0];
			yi=v[3]*x+v[4]*y+m[1];
			val=tex2D(textureImage,xi,yi); //I can not use tex2D with a 3D binded texture
		}
		*/


		boxCUDA[tid]=val;//coalescent access to save global memory

		//box statistics
		meanBox[threadIdx.x]=val;
		stdBox[threadIdx.x]=val*val;
	}

	__syncthreads();
	//add up the box statistics for each block
	int aux = blockDim.x/2;
	int aux2 = threadIdx.x;
	while (aux != 0) 
	{
		if (aux2 < aux)
		{
			stdBox[aux2] += stdBox[aux2 + aux];
			meanBox[aux2] += meanBox[aux2 + aux];
		}
		__syncthreads();
		aux /= 2;
	}

	if (aux2 == 0)
	{
		stdBoxCUDA[blockIdx.x] = stdBox[0];
		meanBoxCUDA[blockIdx.x] = meanBox[0];
	}
}
//==============================end of interpolation kernel===============================================

__global__ void __launch_bounds__(MAX_THREADS_CUDA) addBoxIntensityStatisticsKernel(float* meanBoxCUDA,float* stdBoxCUDA,float* meanFinalCUDA,float* stdFinalCUDA,int boxSize)
{
	__shared__ float meanFinal[sizeMeanStdBoxVector];
	__shared__ float stdFinal[sizeMeanStdBoxVector];

	//copy values
	meanFinal[threadIdx.x] = meanBoxCUDA[threadIdx.x];
	stdFinal[threadIdx.x] = stdBoxCUDA[threadIdx.x];

	__syncthreads();

	//add up the box statistics for each block
	int aux = blockDim.x/2;
	int aux2 = threadIdx.x;
	while (aux != 0) 
	{
		if (aux2 < aux)
		{
			stdFinal[aux2] += stdFinal[aux2 + aux];
			meanFinal[aux2] += meanFinal[aux2 + aux];
		}
		__syncthreads();
		aux /= 2;
	}

	if (aux2 == 0)
	{
		aux = boxSize-1;
		float auxS = meanFinal[0] / (float)boxSize;
		meanFinalCUDA[0] = auxS;
		auxS = stdFinal[0]/((float)aux)-auxS*auxS*((float)boxSize)/((float)aux);
		if(auxS < 1e-3)
			stdFinalCUDA[0] = 1.0;//if the Gausian is too small we might have nothing
		else
			stdFinalCUDA[0] = sqrt(auxS);
	}
}

//=========================beginning of box intensity statistics kernel=====================================
__global__ void __launch_bounds__(MAX_THREADS_CUDA) normalizeBoxKernel(float* boxCUDA,int boxSizeAux,float* meanFinalCUDA,float* stdFinalCUDA)
{
	__shared__ float mean;
	__shared__ float std;

	if(threadIdx.x==0)
		mean = meanFinalCUDA[0];
	if(threadIdx.x==1)
		std = stdFinalCUDA[0];

	__syncthreads();

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(tid<boxSizeAux)
	{
		boxCUDA[tid]=(boxCUDA[tid]-mean)/std;
	}

}

//=========================end of box inetnsity statistics kernel =====================================

//=========================beginning of kernel to calculate cell idx for each voxel in a box================
__global__ void __launch_bounds__(MAX_THREADS_CUDA) calculateCellIdx3DKernel(double* dCUDA,int radiusBox_0, int radiusBox_1, int radiusBox_2,int ellipseIdx, int numEllipsoids,unsigned short int* boxCellIdxCUDA)
{
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	__shared__ float d[dimsImage];//eigenvalues
	__shared__ int boxSize;
	
	//read from global memory to shared memory: unfortunately we do not have coalescent access
	if(threadIdx.x < dimsImage)
	{
		d[threadIdx.x] = dCUDA[threadIdx.x*numEllipsoids+ellipseIdx];
	}
	if(threadIdx.x == dimsImage)
		boxSize = (1+2*radiusBox_0)*(1+2*radiusBox_1)*(1+2*radiusBox_2);

	__syncthreads();


	//calculate interpolation
	if(tid<boxSize)
	{		
		float x,y,z;
		int aux,aux2;
		float rr;
		double xyz[dimsImage];
		long long int auxIdx;

		//calculate x,y,z coordinates for the point
		aux=(1+2*radiusBox_0);
		x=(float)(tid%aux);
		aux2=(tid-(int)x)/aux;
		aux=(1+2*radiusBox_1);
		y=(float)(aux2%aux);
		z=(float)((aux2-(int)y)/aux);
		//recenter and recover anysotropy from teh Gaussian
		x=sqrt(d[0])*(x-radiusBox_0);
		y=sqrt(d[1])*(y-radiusBox_1);
		z=sqrt(d[2])*(z-radiusBox_2);

		//calculate radius
		rr=sqrt((float)(x*x+y*y+z*z));

		//calculate normalized and centered (on a sphere) coordinates
		xyz[0] = x/rr;
		xyz[1] = y/rr;
		xyz[2] = z/rr;


		aux = 1;
		aux << numAngCells;//equivalent to (long long int)pow(2.0f,numAngCells) . Cuda 5.0 release compiled version does not return correct answer with pow function
		if(rr<1e-3)//otherwise it accesses out of bounds memory
			auxIdx = 1;
		else
			vec2pix_nest(aux, xyz, &auxIdx);

		//decide which cell belongs to
		if(rr>=r0CUDA[0]+(numRadialCells-1)*kSigmaCUDA[0])
			auxIdx = 0;//outside the largest ellipsoid
		else if(rr < r0CUDA[0])
			auxIdx = 1;//central ellipsoid
		else{
			aux = (int)floor((rr-r0CUDA[0])/kSigmaCUDA[0]);
			auxIdx = 2 + numCellsHEALPixCUDA[0]*aux + auxIdx;  
		}

		__syncthreads();//to guarantee coalescent access to memory
		//copy data
		boxCellIdxCUDA[tid] = auxIdx;		
	}
}

//=======================================================================================================
//each grop of threads within a block calculates the values of convolution in a whole column along the direction of the separable kernel. It does not provide coalescent memory access along Y and Z axis but we can use the shared memory to store all the column at once
__global__ void __launch_bounds__(MAX_THREADS_CUDA) separableConvolutionKernel(float* volCUDA,int dims_0,int dims_1,int dims_2,int kernelRadius,int dim)
{
	
	__shared__ int kernelSize;

	__shared__ float colConv[maxDiameterBox];
	__shared__ float colOrig[maxDiameterBox];//so we do not overwrite values while computing convolution

	__shared__ int dimsShared[3];//just we can write the code with for loops

	if(threadIdx.x == 0)
		dimsShared [threadIdx.x] = dims_0;
	else if(threadIdx.x == 1)
		dimsShared [threadIdx.x] = dims_1;
	else if(threadIdx.x == 2)
		dimsShared [threadIdx.x] = dims_2;
	else if(threadIdx.x == 3)
		kernelSize = 1+2*kernelRadius;
	__syncthreads();

	
	int iniPosXYZ [dimsImage];
	int offset = (dim+1)%dimsImage;
	iniPosXYZ[dim] = 0; //the initial pos is a plane of dim-th dimension = 0
	iniPosXYZ[offset] = blockIdx.x % dimsShared[offset];// generate a grid withthe other dimensions
	iniPosXYZ[(dim+2)%dimsImage] = (blockIdx.x -iniPosXYZ[offset])/ dimsShared[offset]; 

	int pos = 0;
	int dd = 1;
	for(int ii=0;ii<dimsImage;ii++)
	{
		pos += dd * iniPosXYZ[ii];
		dd *= dimsShared[ii];
	}

	offset = 1;//how much do we have to skip to find next value along the separable kernel axis
	for(int ii=0;ii<dim;ii++)//for ii=0->offset=1->coalescent memory access
		offset *= dimsShared[ii];

	pos += offset*threadIdx.x;//add offset for each particular thread
	//reset values	
	colOrig[threadIdx.x] = volCUDA[pos];
	colConv[threadIdx.x] = 0;

	__syncthreads();

	//calculate value of each pixel
	offset = threadIdx.x - kernelRadius;
	float auxVal;
	for(int ii=0;ii<kernelSize;ii++)
	{
		if(offset<0) //make sure we are within bounds. TODO: allow different boundary conditions. Right no we are extending the last value.
			auxVal = colOrig[0];
		else if(offset>= dimsShared[dim])
			auxVal = colOrig[dimsShared[dim]-1];
		else
			auxVal = colOrig[offset];
		
		colConv[threadIdx.x] += auxVal * kernelSeparableConv[ii];
		
		offset++;
	}

	__syncthreads();

	//copy result back to global memory
	volCUDA[pos] = colConv[threadIdx.x];

}
//======================================================================================================

//========================kernel to calculate the average intensity in each sector======================
__global__ void __launch_bounds__(MAX_THREADS_CUDA) countIntensityPerCellKernel(float* fCellVecCUDA,int* nCellVecCUDA,float* boxCUDA,unsigned short int* boxCellIdxCUDA,int numCellsIdx,int boxSize)
{
	__shared__ float fCellVecCUDAshared[MAX_NUM_CELLS_IDX];
	__shared__ int nCellVecCUDAshared[MAX_NUM_CELLS_IDX];

	//reset values
	if(threadIdx.x<numCellsIdx)
	{
		fCellVecCUDAshared[threadIdx.x] = 0.0f;
		nCellVecCUDAshared[threadIdx.x] = 0;
	}

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid<boxSize)
	{
		unsigned short int nAux = boxCellIdxCUDA[tid];
		if(nAux>0)//we do not care about outside voxels label as zero cell index (we save a lot of atomic additions)
		{
			float fAux = boxCUDA[tid];
			atomicAdd(&(fCellVecCUDAshared[nAux]), fAux);
			atomicAdd(&(nCellVecCUDAshared[nAux]), 1);
		}
	}
	__syncthreads();

	//add to the global variable memory
	if(threadIdx.x<numCellsIdx)
	{
		if(nCellVecCUDAshared[threadIdx.x]>0)
		{
			atomicAdd(&(fCellVecCUDA[threadIdx.x]), fCellVecCUDAshared[threadIdx.x]);
			atomicAdd(&(nCellVecCUDA[threadIdx.x]), nCellVecCUDAshared[threadIdx.x]);
		}
	}
}

//====================end of kernel to calculate the average intensity in each sector======================

//========================================================================================================
void calculateSeparableConvolutionBoxInCUDA(float* volCUDA,const int* volRadiusDims,const double* d,hipStream_t &stream)
{
	const int KsigmaDoG = 5;//kSigma*sigma defines the length of the Gaussian kernel
	int volDims[dimsImage];
	for(int ii=0;ii<dimsImage;ii++)
		volDims[ii] = 1+2*volRadiusDims[ii];

	if(maxDiameterBox>MAX_THREADS_CUDA)
	{
		cout<<"ERROR: at calculateSeparableConvolutionBoxInCUDA. Box cannot be larger than MAX_THREADS_CUDA for this particular implementation of convolution"<<endl;
		exit(3);
	}

	if(maxDiameterBox*maxDiameterBox>MAX_BLOCKS_CUDA)
	{
		cout<<"ERROR: at calculateSeparableConvolutionBoxInCUDA. Box cannot be larger than MAX_BLOCKS_CUDA for this particular implementation of convolution"<<endl;
		exit(3);
	}

	int volSize = volDims[0];
	for(int ii=1;ii<dimsImage;ii++)
		volSize *= volDims[ii];


	int kernelRadius=0;
	float sigmaDoG = 0.0f, diameter = 0.0f;
	float* kernelHOST = new float[1+2*maxRadiusBox];//this is the maximum size for the kernel
	float w = 0.0f;
	//calculate separable convolution for the first sigma
	for (int ii=0;ii<dimsImage;ii++)
	{
		//calculate kernel
		diameter=constDoG_0*sqrt(1.0/d[ii]);//usually when we plot Gaussians they are between 2-3 sigmas
		sigmaDoG=std::max(diameter/constDoG_1,1.0f);
		sigmaDoG=std::min(sigmaDoG,(floor(float(maxRadiusBox)/((float)KsigmaDoG*1.6f))));
		
		kernelRadius = (int)ceil(KsigmaDoG * sigmaDoG);
		w = 0.0f;
		for(int jj=0;jj<1+2*kernelRadius;jj++)
		{
			kernelHOST[jj] = exp(-0.5f*pow((jj-kernelRadius)/sigmaDoG,2));
			w += kernelHOST[jj];
		}
		for(int jj=0;jj<1+2*kernelRadius;jj++)//normalize
			kernelHOST[jj] /= w;
		//copy kernel to constant memory
		HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(kernelSeparableConv),kernelHOST,sizeof(float)*(1+2*kernelRadius)));
		//calculate convolution
		separableConvolutionKernel<<<volSize/volDims[ii],volDims[ii],0,stream>>>(volCUDA,volDims[0],volDims[1],volDims[2],kernelRadius,ii);HANDLE_ERROR_KERNEL;
	}
	//calculate separable convolution for the second sigma
	for (int ii=0;ii<dimsImage;ii++)
	{
		//calculate kernel
		diameter=constDoG_0*sqrt(1.0/d[ii]);//usually when we plot Gaussians they are between 2-3 sigmas
		sigmaDoG=std::max(diameter/constDoG_1,1.0f);
		sigmaDoG=std::min(sigmaDoG,(floor(float(maxRadiusBox)/((float)KsigmaDoG*1.6f))));
		sigmaDoG *=1.6f;//for the DoG
		
		kernelRadius = (int)ceil(KsigmaDoG * sigmaDoG);
		w = 0.0f;
		for(int jj=0;jj<1+2*kernelRadius;jj++)
		{
			kernelHOST[jj] = exp(-0.5f*pow((jj-kernelRadius)/sigmaDoG,2));
			w += kernelHOST[jj];
		}
		for(int jj=0;jj<1+2*kernelRadius;jj++)//normalize
			kernelHOST[jj] /= (-w);//negative so we achieve the DoG effect
		//copy kernel to constant memory
		HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(kernelSeparableConv),kernelHOST,sizeof(float)*(1+2*kernelRadius)));
		//calculate convolution
		separableConvolutionKernel<<<volSize/volDims[ii],volDims[ii],0,stream>>>(volCUDA,volDims[0],volDims[1],volDims[2],kernelRadius,ii);HANDLE_ERROR_KERNEL;
	}

	delete[] kernelHOST;
}

//---------------------------------------------------------------------------------------------------------------------
struct slicePts
{
	double m_z;
	int idx;

	friend bool operator< ( const slicePts& lhs, const slicePts& rhs);
};

inline bool operator< (const slicePts& lhs, const slicePts& rhs)
{	
	return lhs.m_z<rhs.m_z;//in order to sort in ascending order
};


basicEllipticalHaarFeatureVector** calculateEllipticalHaarFeatures(const double *m,const double *W,int numEllipsoids,const imageType *im,const long long int *dims,int devCUDA,int symmetry)
{
	HANDLE_ERROR( hipSetDevice( devCUDA ) );

	size_t freeGPUmem = 0, totalGPUmem = 0;
	HANDLE_ERROR( hipMemGetInfo( &freeGPUmem, &totalGPUmem) ); 
	freeGPUmem -= 3 * 104857600;//We leave 300MB as a safety margin 

	//calculate total memory needed in the GPU
	long long int imSize = dims[0];
	int maxBoxSize = maxDiameterBox;
	for(int ii=1;ii<dimsImage;ii++)
	{
		imSize *= dims[ii];
		maxBoxSize *= maxDiameterBox;
	}

	const size_t sliceOffset = dims[0] * dims[1];
	const size_t sliceMem = sliceOffset * sizeof(imageType) ;
	const size_t boxGPUmem = maxBoxSize * (sizeof(float) + sizeof(unsigned short int)) + (sizeMeanStdBoxVector + 1) * 2 * sizeof(float) + numEllipsoids*dimsImage*sizeof(double) * (1 + (dimsImage+1)/2 + 1 + dimsImage) + numCellsIdx * 2 * (sizeof(float) + sizeof(int));//memory for auxiliary variables
	const size_t neededGPUmem = imSize * sizeof( imageType ) + boxGPUmem;


	if( neededGPUmem < freeGPUmem )//we do not need to partition the image into blocks to fit in memory
	{
		return calculateEllipticalHaarFeaturesBlockWise(m, W, numEllipsoids, im, dims, symmetry);
	}else{//partition the image into blocks along Z axis

		//preallocate memory for block variables
		double *mBlock = new double[numEllipsoids * dimsImage];//upper bound
		double *WBlock = new double[numEllipsoids * dimsImage * (1+ dimsImage) / 2];//upper bound
		int numEllipsoidsBlock;
		long long int dimsBlock[dimsImage];
		dimsBlock[0] = dims[0];
		dimsBlock[1] = dims[1];

		//decide number of slices per block		
		long long int maxSlicesPerBlock = floor( (float(freeGPUmem) - float(boxGPUmem)) / float(sliceMem) );

		if( maxSlicesPerBlock <= maxDiameterBox + 1 )
		{
			cout<<"ERROR: calculateEllipticalHaarFeatures: GPU does not have enough memory to perform feature computation per blocks. maxSlicesPerBlock = "<<maxSlicesPerBlock<<endl;
			cout<<"Total GPU mem = "<<totalGPUmem<<";Free GPU mem = "<<freeGPUmem<<";Box GPU mem = "<<boxGPUmem<<"; Bytes per slice = "<<sliceMem<<endl;
			return NULL;
		}

		//preallocate final result
		basicEllipticalHaarFeatureVector **f = new basicEllipticalHaarFeatureVector*[numEllipsoids];
		for(int ii=0;ii<numEllipsoids;ii++)
			f[ii] = new basicEllipticalHaarFeatureVector;

		//sort points according to Z
		std::vector<slicePts> slicePtsVec(numEllipsoids);
		for(int ii = 0 ; ii < numEllipsoids; ii++)
		{
			slicePtsVec[ii].idx = ii;
			slicePtsVec[ii].m_z = m[numEllipsoids + numEllipsoids + ii];
		}
		std::sort(slicePtsVec.begin(), slicePtsVec.end());

		//calculate elliptical Haar features per block
		long long int iniSlice = 0, endSlice = std::min(maxSlicesPerBlock - 1, dims[2]-1);//endSlice is included
		float ubZ = endSlice - maxRadiusBox;//upper and lower bound for Z coordinates for be included
		if( endSlice == dims[2] - 1)
			ubZ = dims[2];
		int slicePtsIdx = 0;//to keep track of which points we included
		while(1)
		{			
			//check how many centroids to include
			numEllipsoidsBlock = 0;
			int slicePtsIdxOld = slicePtsIdx;
			while( slicePtsIdx < numEllipsoids && slicePtsVec[slicePtsIdx].m_z <= ubZ )
			{
				numEllipsoidsBlock++;
				slicePtsIdx++;
			}
			//copy centroids and precision matrix
			for( int ii = 0; ii < dimsImage; ii++)
			{
				for(int kk = 0; kk <numEllipsoidsBlock; kk++)
				{
					mBlock[ numEllipsoidsBlock * ii + kk] = m[slicePtsVec[slicePtsIdxOld + kk].idx + ii * numEllipsoids];
				}
			}
			const int auxOffset = dimsImage * (1+dimsImage) / 2;
			for( int ii = 0; ii < auxOffset; ii++)
			{
				for(int kk = 0; kk <numEllipsoidsBlock; kk++)
				{
					WBlock[ numEllipsoidsBlock * ii + kk] = W[slicePtsVec[slicePtsIdxOld + kk].idx + ii * numEllipsoids];
				}
			}			
			//offset Z coordinates for a block
			for(int kk = 0; kk <numEllipsoidsBlock; kk++)
			{
				mBlock[ 2 * numEllipsoidsBlock + kk] -= iniSlice;
			}

			//alculate features for this block
			dimsBlock[2] = endSlice - iniSlice + 1;
			basicEllipticalHaarFeatureVector** fBlock = calculateEllipticalHaarFeaturesBlockWise( mBlock,WBlock, numEllipsoidsBlock, &(im[sliceOffset * iniSlice]), dimsBlock, symmetry);
			//parse block results
			if( numEllipsoidsBlock > 0 )
			{
				if( fBlock == NULL )
				{	//error
					for(int ii=0;ii<numEllipsoids;ii++)
					{
						delete f[ii];
					}
					delete[] f;
					f = NULL;
					break;
				}else{//copy results to final vector
					for(int ii=0;ii<numEllipsoidsBlock;ii++)
					{
						(*(f[slicePtsVec[slicePtsIdxOld + ii].idx])) = (*(fBlock[ii]));
						delete fBlock[ii];
					}
					delete[] fBlock;
				}
			}
			//update block limits
			if( endSlice >= dims[2] - 1 )
				break;

			iniSlice = endSlice - 2 * maxRadiusBox - 1;
			endSlice = iniSlice + maxSlicesPerBlock - 1;
			if( endSlice >= dims[2]-1 )//we reached the end
			{
				endSlice = dims[2] - 1;
				ubZ = dims[2];
			}else{
				ubZ = endSlice - maxRadiusBox;
			}
		}

		//release memory
		delete[] mBlock;
		delete[] WBlock;
		slicePtsVec.clear();

		return f;
	}
}

//---------------------------------------------------------------------------

basicEllipticalHaarFeatureVector** calculateEllipticalHaarFeaturesBlockWise(const double *m,const double *W,int numEllipsoids,const imageType *im,const long long int *dims, int symmetry)
{
	basicEllipticalHaarFeatureVector **f=NULL;//vector containing the final value fo features
	

	if(sizeMeanStdBoxVector>MAX_THREADS_CUDA)
	{
		cout<<"ERROR: sizeMeanStdBoxVecotr cannot be bigger than MAX_THREADS_CUDA"<<endl;
		return f;
	}
	if(MAX_NUM_CELLS_IDX<numCellsIdx)
	{
		cout<<"ERROR: MAX_NUM_CELLS_IDX < numCellsIdx. Chang ethe maximum size so we can allocate share memory for reduction purposes"<<endl;
		return f;
	}
	if( numEllipsoids <= 0 )
		return f;

	//---------------allocate memory in GPU and transfer the data----------------------------------------------
	//cout<<"Allocating memory in devCUDA="<<devCUDA<<endl;

	//initialize constant memory
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(r0CUDA),&r0,sizeof(double)));
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(kSigmaCUDA),&kSigma,sizeof(double)));
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(numCellsHEALPixCUDA),&numCellsHEALPix,sizeof(int)));

	int x2pixHOST[128];//HEALPIX constants
	int y2pixHOST[128];
	mk_xy2pix(x2pixHOST,y2pixHOST);
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(x2pix),x2pixHOST,128*sizeof(int)));
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(y2pix),y2pixHOST,128*sizeof(int)));

	//imageType *imCUDA = NULL;
	hipArray* imCUDA = NULL;

	double *mCUDA = NULL;//mean
	double *wCUDA = NULL;//covariance
	
	double *dCUDA = NULL; //eigenvalues
	double *vCUDA = NULL; //eigenvectors

	//boxes are not declared as hipArray (or binded to texture) because we cannot write on cudaArrays
	float *boxCUDA = NULL;//holds the interpolated box for each Gaussian
	float* stdBoxCUDA = NULL;//partial sums of intensity^2 values to normalize box
	float* meanBoxCUDA = NULL;//partial sums of intensity values to normalize box
	float* meanFinalCUDA = NULL;//stores final value for mean
	float* stdFinalCUDA = NULL;//stores final value for mean
	unsigned short int* boxCellIdxCUDA = NULL;//contains the cell id for each voxel, so we can calculate features

	long long int imSize = dims[0];
	int maxBoxSize = maxDiameterBox;
	for(int ii=1;ii<dimsImage;ii++)
	{
		imSize *= dims[ii];
		maxBoxSize *= maxDiameterBox;
	}
	//int maxDiameterBox=2*maxRadiusBox+1;

	//allocate cuda array for image and for box(texture memory)
	// allocate CudaArray
	hipChannelFormatDesc channelDescImage = hipCreateChannelDesc<imageType>();
	
	
	//create a hipExtent structure, storing the dimensions of the 3D texture
	hipExtent imageSize = make_hipExtent(dims[0], dims[1], dims[2]);
	

	if(dimsImage==3)
	{
		HANDLE_ERROR(hipMalloc3DArray(&imCUDA, &channelDescImage, imageSize));
	
		/*create a hipMemcpy3DParms structures--this is a structure that tells cuda how to copy data using the hipMemcpy3D function. Basically, this prevents having to pass a bunch of parameters in favor of a single complex structure.*/
		hipMemcpy3DParms copyParmsImage={0};
		copyParmsImage.srcPtr = make_hipPitchedPtr((void*)im, dims[0]*sizeof(imageType), dims[0], dims[1]); 
		copyParmsImage.dstArray = imCUDA;
		copyParmsImage.extent = imageSize;
		copyParmsImage.kind = hipMemcpyHostToDevice;
		HANDLE_ERROR( hipMemcpy3D(&copyParmsImage));
		
	}else{//TODO: do it for 2D;
		cout<<"ERROR: code is not ready for dimsImage="<<dimsImage<<endl;
		return f;
	}
	
	//set the parameters for the global texture variable
	textureImage.normalized = false; //coordinates are not between [0,1]^dimsImage but between image size boundaries
	textureImage.filterMode = hipFilterModePoint;//nearest neighbor interpolation. Use hipFilterModeLinear for linear interpolation
	textureImage.addressMode[0] = hipAddressModeClamp;//How out of bounds requests are handled. For non-normalized mode only clamp is supported. In clamp addressing mode x is replaced by 0 if x<0 and N-1 if x>=N;
	textureImage.addressMode[1] = hipAddressModeClamp;
	textureImage.addressMode[2] = hipAddressModeClamp;

	
	//bind the texture to the array
	HANDLE_ERROR( hipBindTextureToArray(textureImage, imCUDA, channelDescImage) );


	//allocate memory to hold the image and boxes (we just allocate the max allowed since it is small enough)	
	//HANDLE_ERROR( hipMalloc( (void**)&(imCUDA), imSize*sizeof(imageType) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(boxCUDA), maxBoxSize*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(boxCellIdxCUDA), maxBoxSize*sizeof(unsigned short int) ) );
	
	//int sizeMeanStdBoxVector = (int)ceil(((float)(maxDiameterBox*maxDiameterBox*maxDiameterBox)/((float)MAX_THREADS_CUDA));
	//we need it ot be a power of 2
	//sizeMeanStdBoxVector = (int)pow(2.0f,(int)ceil(log2((float)sizeMeanStdBoxVector)));
	
	HANDLE_ERROR( hipMalloc( (void**)&(meanBoxCUDA), sizeMeanStdBoxVector*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(stdBoxCUDA), sizeMeanStdBoxVector*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(meanFinalCUDA), sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(stdFinalCUDA), sizeof(float) ) );
	

	//allocate memory for Gaussian centroids and covariance
	HANDLE_ERROR( hipMalloc( (void**)&(mCUDA), numEllipsoids*dimsImage*sizeof(double) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(wCUDA), numEllipsoids*dimsImage*((dimsImage+1)/2)*sizeof(double) ) );

	//copy to GPU
	//HANDLE_ERROR( hipMemcpy( imCUDA, im, imSize*sizeof(imageType) , hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( mCUDA, m, numEllipsoids*dimsImage*sizeof(double) , hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( wCUDA, W, numEllipsoids*dimsImage*((dimsImage+1)/2)*sizeof(double) , hipMemcpyHostToDevice ) );

	//allocate memory for eigenvalues and eigenvectors
	HANDLE_ERROR( hipMalloc( (void**)&(dCUDA), numEllipsoids*dimsImage*sizeof(double) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(vCUDA), numEllipsoids*dimsImage*dimsImage*sizeof(double) ) );

	//generate strems to parallelize kernel and data transfer
	hipStream_t stream0, stream1;
	HANDLE_ERROR( hipStreamCreate( &stream0));
	HANDLE_ERROR( hipStreamCreate( &stream1));

	float* fCellVecHOST = NULL;//stores the sum of intensities for each cell
	int* nCellVecHOST = NULL;//stores the weights so we can calculate average
	float* fCellVecHOSTDoG = NULL;//stores the sum of intensities for each cell
	int* nCellVecHOSTDoG = NULL;//stores the weights so we can calculate average

	float* fCellVecCUDA = NULL;//stores the sum of intensities for each cell
	int* nCellVecCUDA = NULL;//stores the weights so we can calculate average
	float* fCellVecCUDADoG = NULL;//stores the sum of intensities for each cell
	int* nCellVecCUDADoG = NULL;//stores the weights so we can calculate average
	//allocate as pinned memory so transfers are faster and we can use streams to launch kernels in parallel with data transfer (Chapter 10 of CUDA By example book)
	HANDLE_ERROR( hipHostAlloc( (void**) &fCellVecHOST, numCellsIdx*sizeof(float),hipHostMallocDefault));
	HANDLE_ERROR( hipHostAlloc( (void**) &nCellVecHOST, numCellsIdx*sizeof(int),hipHostMallocDefault));
	HANDLE_ERROR( hipMalloc( (void**)&(fCellVecCUDA), numCellsIdx*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(nCellVecCUDA), numCellsIdx*sizeof(int)) );
	HANDLE_ERROR( hipHostAlloc( (void**) &fCellVecHOSTDoG, numCellsIdx*sizeof(float),hipHostMallocDefault));
	HANDLE_ERROR( hipHostAlloc( (void**) &nCellVecHOSTDoG, numCellsIdx*sizeof(int),hipHostMallocDefault));
	HANDLE_ERROR( hipMalloc( (void**)&(fCellVecCUDADoG), numCellsIdx*sizeof(float) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(nCellVecCUDADoG), numCellsIdx*sizeof(int)) );

	//----------------calculate eigenvalues and eigenvectors for each Gaussian---------------
	int numThreads=std::min(MAX_THREADS_CUDA,numEllipsoids);
	int numBlocks=std::min(MAX_BLOCKS_CUDA,(numEllipsoids+numThreads-1)/numThreads);
	computeEigenDecompositionKernel<<<numBlocks,numThreads>>>(wCUDA,numEllipsoids,dCUDA,vCUDA,1);HANDLE_ERROR_KERNEL;//egigenvalues organized in ascend order
	//copy info back so we know dimensions for each box
	double *dHOST = new double[numEllipsoids*dimsImage];
	double *vHOST = new double[numEllipsoids*dimsImage*dimsImage];

	//apply symmetry if requested by user to artifically extend training data
	if(symmetry>7)
	{
		cout<<"ERROR: symmetry input cannot be higher than 7"<<endl;
		return f;
	}else if(symmetry>0){//apply symmetry before copying eigenvalues to host
		numThreads=std::min(MAX_THREADS_CUDA,numEllipsoids*dimsImage*dimsImage);
		numBlocks=std::min(MAX_BLOCKS_CUDA,(numEllipsoids*dimsImage*dimsImage+numThreads-1)/numThreads);
		applySymmetryToEigenvectorsKernel<<<numBlocks,numThreads>>>(vCUDA,numEllipsoids,symmetry);HANDLE_ERROR_KERNEL;//egigenvalues organized in ascend order
	}

	HANDLE_ERROR(hipMemcpy(dHOST,dCUDA,sizeof(double)*numEllipsoids*dimsImage,hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(vHOST,vCUDA,sizeof(double)*numEllipsoids*dimsImage*dimsImage,hipMemcpyDeviceToHost));

	//------------------------------debug-----------------
	/*
	printf("===============DEBUGGING: Checking for NaN in eigenvalues for %d ellipsoids\n", numEllipsoids);
	int numNan = 0;
	for (int ii = 0; ii < numEllipsoids*dimsImage; ii++)
	{
		if (std::isnan(dHOST[ii]))
		{
			numNan++;
			int vv = ii % numEllipsoids;
			printf("ii = %d; W = %f %f %f %f %f %f\n", vv, W[vv], W[vv + 1 * numEllipsoids], W[vv + 2 * numEllipsoids], W[vv + 3 * numEllipsoids], W[vv + 4 * numEllipsoids], W[vv + 5 * numEllipsoids]);
		}
	}
	if (numNan > 0)
		exit(3);
	*/
	//---------------------------------------------------

	//----------main for loop calculating features for each ellipsoid---------------------
	//cout<<"Starting main loop to calculate features for "<<numEllipsoids<<" boxes"<<endl;
	int radiusBox[dimsImage];
	int pos = 0;
	const double scaleSigma = r0+numRadialCells*kSigma;
	//allocate main memory for output
	f = new basicEllipticalHaarFeatureVector*[numEllipsoids];
	for(int ii=0;ii<numEllipsoids;ii++)
		f[ii] = new basicEllipticalHaarFeatureVector;
	
	//verify dimsImage -- was previously nested within for loop below, and is superfluous because of 2D check above
	/*
	if(dimsImage==2)
	{
		//TODO: I cannot define a 2D and 3D texture atthe same time->I need to write separate codes or thing how to do it
		//interpolate2DBoxKernel<<<numBlocks,numThreads>>>(mCUDA,vCUDA,ii,numEllipsoids,boxCUDA,radiusBox[0],radiusBox[1],meanBoxCUDA,stdBoxCUDA);
		//HANDLE_ERROR_KERNEL;
		cout<<"ERROR: code is not ready for 2D interpolation"<<endl;
		for(int ii=0;ii<numEllipsoids;ii++) delete f[ii];
		delete[] f;
		f = NULL;
		return f;
	}*/
	
	unsigned int dHOST_index;
	for(int ii=0;ii<numEllipsoids;ii++)
	{

		//verify eigenvalues before proceeding with this ellipsoid
		double dAux[dimsImage];
		bool fail_this_ellipsoid = false;
		for(int jj=0;jj<dimsImage;jj++) {
			dHOST_index = ii + numEllipsoids *jj;
			if ( isnan(dHOST[dHOST_index]) || dHOST[dHOST_index] <= 0 ) {
				fail_this_ellipsoid = true;
				break;
			}
			dAux[jj] = dHOST[dHOST_index];//select eigenvalues
		}
		if (fail_this_ellipsoid) {
			continue;
		}
		
		//reset box statistics
		HANDLE_ERROR(hipMemset(meanBoxCUDA,0,sizeof(float)*sizeMeanStdBoxVector));
		HANDLE_ERROR(hipMemset(stdBoxCUDA,0,sizeof(float)*sizeMeanStdBoxVector));

		//reset cell idx and values before streams
		HANDLE_ERROR(hipMemset(fCellVecCUDA,0,numCellsIdx*sizeof(float)));
		HANDLE_ERROR(hipMemset(nCellVecCUDA,0,numCellsIdx*sizeof(int)));
		HANDLE_ERROR(hipMemset(fCellVecCUDADoG,0,numCellsIdx*sizeof(float)));
		HANDLE_ERROR(hipMemset(nCellVecCUDADoG,0,numCellsIdx*sizeof(int)));

		

		//calculate rotated box (interpolation)
		pos = ii;
	    int boxSizeAux = 1;
		for(int jj=0;jj<dimsImage;jj++)
		{
			radiusBox[jj] = (int)ceil(scaleSigma*sqrt(1.0/dHOST[pos]));
			radiusBox[jj] = std::max(radiusBox[jj],minRadiusBox);
			radiusBox[jj] = std::min(radiusBox[jj],maxRadiusBox);
			pos+=numEllipsoids;
			boxSizeAux *= (1+2*radiusBox[jj]);
		}
		numThreads=MAX_THREADS_CUDA; //we do reduction (i.e. we need a power of 2) and most of the time boxSizeAux>MAX_THREADS_CUDA, so we just set it to MAX_THREADS_CUDA
		numBlocks=std::min(MAX_BLOCKS_CUDA,(boxSizeAux+numThreads-1)/numThreads);
		
		//if(dimsImage==3) -- this is verified above in this function
		//{
			interpolate3DBoxKernel<<<numBlocks,numThreads>>>(mCUDA,vCUDA,ii,numEllipsoids,boxCUDA,radiusBox[0],radiusBox[1],radiusBox[2],meanBoxCUDA,stdBoxCUDA);
			HANDLE_ERROR_KERNEL;
		//}

		//------------------------------------------------------------------------
		//normalize the box using the partial sums calculated during interpolation
		addBoxIntensityStatisticsKernel<<<1,sizeMeanStdBoxVector>>>(meanBoxCUDA,stdBoxCUDA,meanFinalCUDA,stdFinalCUDA,boxSizeAux);HANDLE_ERROR_KERNEL;
		normalizeBoxKernel<<<numBlocks,numThreads>>>(boxCUDA,boxSizeAux,meanFinalCUDA,stdFinalCUDA);HANDLE_ERROR_KERNEL;
		

		//----------------------------------------------------
		//calculate basic features values (sectors) for the box
		//figure out the cell for each voxel in the box
		calculateCellIdx3DKernel<<<numBlocks,numThreads>>>(dCUDA,radiusBox[0], radiusBox[1], radiusBox[2],ii, numEllipsoids,boxCellIdxCUDA);HANDLE_ERROR_KERNEL;
		
		
		//make sure GPU finishes before we launch two different streams
		HANDLE_ERROR(hipDeviceSynchronize());	
		


		//---------------------------debug wirte out all boxes----------------------------------------
		/*
		char bufferD[256];		
		sprintf(bufferD, "G:/temp/debugTGMM/box_%.6d.raw", countBoxesDebug);
		countBoxesDebug++;
		string filenameDebug(bufferD);
		cout << "DEBUGGING: ELLIPTICAL HAAR FEATURES saving box to " << filenameDebug.c_str() << " " << (1 + 2 * radiusBox[0]) << "," << (1 + 2 * radiusBox[1]) << "," << (1 + 2 * radiusBox[2]);
		cout << ";countBox = " << countBoxesDebug << "; ii = " << ii << "; m = " << m[ii + 0 * numEllipsoids] << ", " << m[ii + 1 * numEllipsoids] << ", " << m[ii + 2 * numEllipsoids] << "; W = " << W[ii + 0 * numEllipsoids] << ", " << W[ii + 1 * numEllipsoids] << ", " << W[ii + 2 * numEllipsoids] <<","<< W[ii + 3 * numEllipsoids] << ", " << W[ii + 4 * numEllipsoids] << ", " << W[ii + 5 * numEllipsoids] << endl;
		float* boxHOST = new float[boxSizeAux];
		HANDLE_ERROR( hipMemcpy( boxHOST, boxCUDA, boxSizeAux*sizeof(float) , hipMemcpyDeviceToHost) );
		FILE* fid=fopen(filenameDebug.c_str(),"wb");
		fwrite(radiusBox, sizeof(int), dimsImage, fid );//to be able to load them in Matlab
		fwrite(boxHOST,sizeof(float),boxSizeAux,fid);
		fclose(fid);
		delete[] boxHOST;
		*/
		//--------------------------------------------------------------------------------------------
		
		//-------------------debug-------------------------------
		/*
		cout<<"DEBUGGING: saving box to E:\\temp\\box.bin with size "<<(1+2*radiusBox[0])<<","<<(1+2*radiusBox[1])<<","<<(1+2*radiusBox[2])<<endl;
		float* boxHOST = new float[boxSizeAux];
		HANDLE_ERROR( hipMemcpy( boxHOST, boxCUDA, boxSizeAux*sizeof(float) , hipMemcpyDeviceToHost) );
		FILE* fid=fopen("E:/temp/box.bin","wb");
		fwrite(boxHOST,sizeof(float),boxSizeAux,fid);
		fclose(fid);
		delete[] boxHOST;

		unsigned short int* boxCellIdxHOST = new unsigned short int[boxSizeAux];
		HANDLE_ERROR( hipMemcpy( boxCellIdxHOST, boxCellIdxCUDA, boxSizeAux*sizeof(unsigned short int) , hipMemcpyDeviceToHost) );
		fid=fopen("E:/temp/boxCellIdx.bin","wb");
		fwrite(boxCellIdxHOST,sizeof(unsigned short int),boxSizeAux,fid);
		fclose(fid);
		delete[] boxCellIdxHOST;
		*/
		//------------------------------------------------------		
		

		//calculate values of f for each cell 
		countIntensityPerCellKernel<<<numBlocks,numThreads,0,stream0>>>(fCellVecCUDA,nCellVecCUDA,boxCUDA,boxCellIdxCUDA,numCellsIdx,boxSizeAux);
		HANDLE_ERROR( hipMemcpyAsync( fCellVecHOST, fCellVecCUDA, numCellsIdx*sizeof(float) , hipMemcpyDeviceToHost, stream0 ) );
		HANDLE_ERROR( hipMemcpyAsync( nCellVecHOST, nCellVecCUDA, numCellsIdx*sizeof(int) , hipMemcpyDeviceToHost, stream0 ) );

		//calculate DoG of the box
		calculateSeparableConvolutionBoxInCUDA(boxCUDA,radiusBox,dAux,stream1);				
		countIntensityPerCellKernel<<<numBlocks,numThreads,0,stream1>>>(fCellVecCUDADoG,nCellVecCUDADoG,boxCUDA,boxCellIdxCUDA,numCellsIdx,boxSizeAux);

		//calculate values of f for each cell 
		HANDLE_ERROR( hipMemcpyAsync( fCellVecHOSTDoG, fCellVecCUDADoG, numCellsIdx*sizeof(float) , hipMemcpyDeviceToHost, stream1 ) );
		HANDLE_ERROR( hipMemcpyAsync( nCellVecHOSTDoG, nCellVecCUDADoG, numCellsIdx*sizeof(int) , hipMemcpyDeviceToHost, stream1 ) );

		
		HANDLE_ERROR( hipStreamSynchronize( stream0));
		HANDLE_ERROR( hipStreamSynchronize( stream1));
		
		
		//----------------------debug-------------------------
		/*
		float* boxHOSTdog = new float[boxSizeAux];
		HANDLE_ERROR( hipMemcpy( boxHOSTdog, boxCUDA, boxSizeAux*sizeof(float) , hipMemcpyDeviceToHost) );
		cout<<"DEBUGGING: saving box to E:\\temp\\boxDoG.bin with size "<<(1+2*radiusBox[0])<<","<<(1+2*radiusBox[1])<<","<<(1+2*radiusBox[2])<<endl;
		fid=fopen("E:/temp/boxDoG.bin","wb");
		fwrite(boxHOSTdog,sizeof(float),boxSizeAux,fid);
		fclose(fid);
		delete [] boxHOSTdog;
		*/
		/*
		cout<<"DEBUGGING: CUDA code elliptical features"<<endl;
		for(int hh = 0; hh<numCellsIdx; hh++)
		{
			cout<<fCellVecHOST[hh]<<" "<<nCellVecHOST[hh]<<" "<<fCellVecHOSTDoG[hh]<<" "<<nCellVecHOSTDoG[hh]<<endl;
		}
		*/
		
		//-----------------------------------------------------
		


		//calculate excentricity 
		int count = 0;
		for(int jj=0;jj<dimsImage;jj++)
			for(int kk=jj+1;kk<dimsImage;kk++)
			{
				if(dAux[kk]<1e-10)
					f[ii]->excentricity[count] = 0.0f;
				else
					f[ii]->excentricity[count] = dAux[jj]/dAux[kk];

				count++;
			}
		//----------------------------------------------------
		//calculate all the cells and rings
		f[ii]->ringAvgIntensity[0] = fCellVecHOST[1] / (float)(nCellVecHOST[1]);//central ring
		f[ii]->ringAvgIntensityDoG[0] = fCellVecHOSTDoG[1] / (float)(nCellVecHOSTDoG[1]);//central ring
		count = 0;		
		int count2 = count + 2;
		for(int jj=0;jj<numRadialCells-1;jj++)
		{
			int Nring = 0, NringDoG = 0;
			float Wring = 0.0f, WringDoG = 0.0f;
			for(int kk=0;kk<numCellsHEALPix;kk++)
			{
				Nring += nCellVecHOST[count2];
				Wring += fCellVecHOST[count2];
				
				if( nCellVecHOST[count2] == 0)//sometimes ellipsois are so flat in Z that some cells do not get any pixel
					f[ii]->cellAvgIntensity[count] = 0;
				else
					f[ii]->cellAvgIntensity[count] = fCellVecHOST[count2] / (float)(nCellVecHOST[count2]);

				NringDoG += nCellVecHOSTDoG[count2];
				WringDoG += fCellVecHOSTDoG[count2];
				if( nCellVecHOSTDoG[count2] == 0)//sometimes ellipsois are so flat in Z that some cells do not get any pixel
					f[ii]->cellAvgIntensityDoG[count] = 0;
				else
					f[ii]->cellAvgIntensityDoG[count] = fCellVecHOSTDoG[count2] / (float)(nCellVecHOSTDoG[count2]);

				count++;
				count2++;
			}
			if( Nring == 0 )
				f[ii]->ringAvgIntensity[jj+1] = 0;
			else
				f[ii]->ringAvgIntensity[jj+1] = Wring/(float)Nring;

			if( NringDoG == 0 )
				f[ii]->ringAvgIntensityDoG[jj+1] = 0;
			else
				f[ii]->ringAvgIntensityDoG[jj+1] = WringDoG/(float)NringDoG;
		}
		//----------------------------------------------------
		//expand the set of features by combining pairs
		//TODO
	}

	//unbind textures
	hipUnbindTexture(textureImage);

	//deallocate memory
	HANDLE_ERROR( hipFree( mCUDA ) );
	HANDLE_ERROR( hipFree( wCUDA ) );
	HANDLE_ERROR( hipFree( dCUDA ) );
	HANDLE_ERROR( hipFree( vCUDA ) );
	HANDLE_ERROR( hipFree( meanBoxCUDA ) );
	HANDLE_ERROR( hipFree( stdBoxCUDA ) );	
	HANDLE_ERROR( hipFree( meanFinalCUDA ) );
	HANDLE_ERROR( hipFree( stdFinalCUDA ) );
	HANDLE_ERROR( hipFree( boxCUDA ) );
	HANDLE_ERROR( hipFree( boxCellIdxCUDA ) );
	//HANDLE_ERROR( hipFree( imCUDA ) );
	HANDLE_ERROR( hipFreeArray( imCUDA ) );
	
	HANDLE_ERROR( hipStreamDestroy( stream0));
	HANDLE_ERROR( hipStreamDestroy( stream1));

	delete[] dHOST;
	delete[] vHOST;
	HANDLE_ERROR( hipHostFree( fCellVecHOST ));
	HANDLE_ERROR( hipHostFree( nCellVecHOST ));
	HANDLE_ERROR( hipFree( fCellVecCUDA));
	HANDLE_ERROR( hipFree( nCellVecCUDA));
	HANDLE_ERROR( hipHostFree( fCellVecHOSTDoG ));
	HANDLE_ERROR( hipHostFree( nCellVecHOSTDoG ));
	HANDLE_ERROR( hipFree( fCellVecCUDADoG));
	HANDLE_ERROR( hipFree( nCellVecCUDADoG));


	return f;
}



/*
	//--------------------------------------debug: cuda3Dtexture------------------------
		cout<<"DEBUGGING: 3D texture access. Writing copied file out"<<endl;
		imageType* imAuxCUDA, *imAuxHOST;
		imAuxHOST = new imageType[imSize];
		HANDLE_ERROR( hipMalloc( (void**)&(imAuxCUDA),imSize*sizeof(imageType) ) );

		int numThreadsAux=std::min(MAX_THREADS_CUDA,(int)imSize);
		int numBlocksAux=std::min(MAX_BLOCKS_CUDA,((int)imSize+numThreadsAux-1)/numThreadsAux);

		debuggingCopy3DTextureKernel<<<numBlocksAux,numThreadsAux>>>(imAuxCUDA,imSize,dims[0],dims[1],dims[2]);HANDLE_ERROR_KERNEL;
		HANDLE_ERROR( hipMemcpy( imAuxHOST, imAuxCUDA, imSize*sizeof(imageType) , hipMemcpyDeviceToHost ) );

		FILE* fim=fopen("E:/temp/imCUDAcopy.bin","wb");
		fwrite(imAuxHOST,sizeof(imageType),imSize,fim);
		fclose(fim);

		fim=fopen("E:/temp/imHOSTcopy.bin","wb");
		fwrite(im,sizeof(imageType),imSize,fim);
		fclose(fim);
		//-------------------------------------------------------------------

		*/

/*
//=======================================debug kernel========================================================
__global__ void __launch_bounds__(MAX_THREADS_CUDA) debuggingCopy3DTextureKernel (imageType* imAuxCUDA,long long int imSize,int dims_0,int dims_1,int dims_2)
{
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	float x,y,z;
	int aux,aux2;
		
	while(tid<imSize)
	{
		

		//calculate x,y,z coordinates for the point
		aux=dims_0;
		x=(float)(tid%aux);
		aux2=(tid-(int)x)/aux;
		aux=dims_1;
		y=(float)(aux2%aux);
		z=(float)((aux2-(int)y)/aux);

		imAuxCUDA[tid] = tex3D(textureImage,x,y,z);

		if(tid==0)
		{
			printf("Value at (%f,%f,%f) from debuggin kernel is %d\n",x,y,z,int(imAuxCUDA[tid]));
		}

		tid+= blockDim.x * gridDim.x;
	}
}

//=========================================================================================
*/


bool basicEllipticalHaarFeatureVector::debugIsAnyFeatureNan()
{
	bool isnan_= false;

	for (int ii = 0; ii < dimsImage; ii++)
		isnan_ = isnan_ || isnan(excentricity[ii]);

	for (int ii = 0; ii<numRings; ii++)
		isnan_ = isnan_ || isnan(ringAvgIntensity[ii]);

	for (int ii = 0; ii<numCells; ii++)
		isnan_ = isnan_ || isnan(cellAvgIntensity[ii] );

	for (int ii = 0; ii<numRings; ii++)
		isnan_ = isnan_ || isnan(ringAvgIntensityDoG[ii]);

	for (int ii = 0; ii<numCells - 1; ii++)
		isnan_ = isnan_ || isnan(cellAvgIntensityDoG[ii]);

	isnan_ = isnan_ || isnan(cellAvgIntensityDoG[numCells - 1]);

	return isnan_;
}
