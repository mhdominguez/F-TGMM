#include "hip/hip_runtime.h"
/*
 * knnCuda.cu
 *
 *  Created on: Jul 15, 2011
 *      Author: amatf
 */

#include "GMEMcommonCUDA.h"
#include "knnCuda_v1.h"
#include "external/book.h"
#include <iostream>
#include <algorithm>

#if defined(_WIN32) || defined(_WIN64)
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#endif

__constant__ float knn_v1_refCUDA[MAX_REF_POINTS*3];
__constant__ float knn_v1_scaleCUDA[dimsImage];

__device__ inline void findMaxPosition(float *distArray,float* minDist,int *pos)
{
	(*minDist)=distArray[0];
	(*pos)=0;
	for(int ii=1;ii<maxGaussiansPerVoxel;ii++) 
	{
		if((*minDist)<distArray[ii])
		{
			(*minDist)=distArray[ii];
			(*pos)=ii;
		}
	}
}

__global__ void __launch_bounds__(MAX_THREADS) knnKernel(int *indCUDA,float *queryCUDA,int ref_nb,long long int query_nb)
{
	// map from threadIdx/BlockIdx to pixel position
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int offset = blockDim.x * gridDim.x;

	if(tid>=query_nb) return;

	//int kMinusOne=maxGaussiansPerVoxel-1;
	float minDist[maxGaussiansPerVoxel];//to mantain distance for each index: since K is very small instead of a priority queue we keep a sorted array
	int indAux[maxGaussiansPerVoxel];
	float queryAux[dimsImage];
	float minDistThr;

	float dist,distAux;
	int jj2,minPos;


	jj2=tid;

	//global memory: organized as x_1,x_2,x_3,....,y_1,y_2,...,z_1,... to have coalescent access
	queryAux[0]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[1]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[2]=queryCUDA[jj2];


	int refIdx=-3;
	for(int jj=0;jj<maxGaussiansPerVoxel;jj++) minDist[jj]=1e32;//equivalent to infinity
	minDistThr=1e32;
	minPos=0;
	for(int ii=0;ii<ref_nb;ii++)
	{
		__syncthreads();//to access constant memory coherently
		refIdx+=3;

		distAux=(queryAux[0]-knn_v1_refCUDA[refIdx])*knn_v1_scaleCUDA[0];
		dist=distAux*distAux;
		if(dist>minDistThr) continue;
		distAux=(queryAux[1]-knn_v1_refCUDA[refIdx+1])*knn_v1_scaleCUDA[1];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;
		distAux=(queryAux[2]-knn_v1_refCUDA[refIdx+2])*knn_v1_scaleCUDA[2];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;

		//insert element" minimize memory exchanges
		minDist[minPos]=dist;
		indAux[minPos]=ii;
		findMaxPosition(minDist,&minDistThr,&minPos);
	}

	__syncthreads();//I need this to have coalescent memory access to inCUDA: speeds up the code by x4

	//copy indexes to global memory
	jj2=tid;
	for(int jj=0;jj<maxGaussiansPerVoxel;jj++)
	{
		//indCUDA[jj+jj2]=indAux[jj];
		indCUDA[jj2]=indAux[jj];
		jj2+=query_nb;
	}
}

//===========================================================================================
__global__ void __launch_bounds__(MAX_THREADS) knnKernelNoConstantMemory(int *indCUDA,float *queryCUDA,float* anchorCUDA,int ref_nb,long long int query_nb)
{
	// map from threadIdx/BlockIdx to pixel position
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int offset = blockDim.x * gridDim.x;

	if(tid>=query_nb) return;

	float minDist[maxGaussiansPerVoxel];//to mantain distance for each index: since K is very small instead of a priority queue we keep a sorted array
	int indAux[maxGaussiansPerVoxel];
	float queryAux[dimsImage];
	float minDistThr;

	float dist,distAux;
	int jj2,minPos;

	jj2=tid;

	//global memory: organized as x_1,x_2,x_3,....,y_1,y_2,...,z_1,... to have coalescent access
	queryAux[0]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[1]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[2]=queryCUDA[jj2];

	int refIdx=-3;
	for(int jj=0;jj<maxGaussiansPerVoxel;jj++) minDist[jj]=1e32;//equivalent to infinity
	minDistThr=1e32;
	minPos=0;
	for(int ii=0;ii<ref_nb;ii++)
	{
		__syncthreads();//to access constant memory coherently
		refIdx+=3;

		distAux=(queryAux[0]-anchorCUDA[refIdx])*knn_v1_scaleCUDA[0];
		dist=distAux*distAux;
		if(dist>minDistThr) continue;
		distAux=(queryAux[1]-anchorCUDA[refIdx+1])*knn_v1_scaleCUDA[1];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;
		distAux=(queryAux[2]-anchorCUDA[refIdx+2])*knn_v1_scaleCUDA[2];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;

		//insert element" minimize memory exchanges
		minDist[minPos]=dist;
		indAux[minPos]=ii;
		findMaxPosition(minDist,&minDistThr,&minPos);
	}

	__syncthreads();//I need this to have coalescent memory access to inCUDA: speeds up the code by x4

	//copy indexes to global memory
	jj2=tid;
	for(int jj=0;jj<maxGaussiansPerVoxel;jj++)
	{
		//indCUDA[jj+jj2]=indAux[jj];
		indCUDA[jj2]=indAux[jj];
		jj2+=query_nb;
	}
}
//===========================================================================================
__device__ inline void Comparator(float& keyA,int& valA,float& keyB,int& valB,unsigned int dir)
{
    float t;
    int v;
    if( (keyA > keyB) == dir ){
        t = keyA; keyA = keyB; keyB = t;
        v = valA; valA = valB; valB = v;
    }
}
//this kernel needs to be called with MAX_THREADS_CUDA
__global__ void __launch_bounds__(MAX_THREADS_CUDA) knnKernelSorting(int *indCUDA,float *queryCUDA,float* anchorCUDA,int ref_nb,long long int query_nb)
{
	
	//Shared memory storage for one or more short vectors
	__shared__ float s_key[MAX_THREADS_CUDA];//distance
	__shared__ int s_val[MAX_THREADS_CUDA];//index
	__shared__ int indAux[maxGaussiansPerVoxel];
	__shared__ float minDist[maxGaussiansPerVoxel];

	float x_n[dimsImage];
	unsigned int dir=0;//ascending order sorting

	// map from threadIdx/BlockIdx to pixel position
	long long int tid = blockIdx.x;
	long long int pos2;
	float dist,aux;

	int maxOffset=((ref_nb+MAX_THREADS_CUDA-1)/MAX_THREADS_CUDA)*MAX_THREADS_CUDA;

	while(tid<query_nb)
	{
		//load query point
		pos2=tid;
		x_n[0]=queryCUDA[pos2];
		pos2+=query_nb;
		x_n[1]=queryCUDA[pos2];
		pos2+=query_nb;
		x_n[2]=queryCUDA[pos2];

		for(int offset=threadIdx.x;offset<maxOffset;offset+=MAX_THREADS_CUDA)
		{
			//calculate distance
			if(offset<ref_nb)
			{
				aux=x_n[0]-anchorCUDA[offset];
				dist=aux*aux;
				offset+=ref_nb;
				aux=x_n[0]-anchorCUDA[offset];
				dist+=aux*aux;
				offset+=ref_nb;
				aux=x_n[2]-anchorCUDA[offset];
				dist+=aux*aux;
			}else{
				dist=1e32;
			}
			s_val[threadIdx.x]=offset;
			s_key[threadIdx.x]=dist;
			__syncthreads();

			//sort value and key in shared memory: bitonc search from Cuda SDK
			for(unsigned int size = 2; size < MAX_THREADS_CUDA; size <<= 1)
			{
				//Bitonic merge
				unsigned int ddd = dir ^ ( (threadIdx.x & (size / 2)) != 0 );
				for(unsigned int stride = size / 2; stride > 0; stride >>= 1)
				{
					__syncthreads();
					unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
					Comparator(s_key[pos +      0], s_val[pos +      0],s_key[pos + stride], s_val[pos + stride],ddd);
				}
			}

			//ddd == dir for the last bitonic merge step
			{
				for(unsigned int stride = MAX_THREADS_CUDA / 2; stride > 0; stride >>= 1){
					__syncthreads();
					unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
					Comparator(s_key[pos +      0], s_val[pos +      0],s_key[pos + stride], s_val[pos + stride],dir);
				}
			}
			__syncthreads();

			//merge this batch of distances with short sorted array
			if(offset<maxGaussiansPerVoxel)//we just need to copy in the first iteration
			{
				indAux[offset]=s_val[offset];
				minDist[offset]=s_key[offset];
			}else
			{

				if(threadIdx.x==0){ //merge two sorted arrays
					int ptr1=0;
					//int	ptr2=0;
					while(ptr1<maxGaussiansPerVoxel)
					{
							ptr1++;
							//TODO: finish this although the kernel is really slow (in comparison) even without this part
					}
				}
				
			}
			__syncthreads();
		}
		//copy indexes to global memory
		if(threadIdx.x<maxGaussiansPerVoxel)
		{
			indCUDA[tid+threadIdx.x*query_nb]=indAux[threadIdx.x];//not coalescence
		}
		//update pointer for next query_point to check
		tid+=gridDim.x;
		__syncthreads();

	}
}

__global__ void __launch_bounds__(MAX_THREADS) knnKernelSortedArray(int *indCUDA,float *queryCUDA,int ref_nb,long long int query_nb)
{
	// map from threadIdx/BlockIdx to pixel position
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int offset = blockDim.x * gridDim.x;

	if(tid>=query_nb) return;

	int kMinusOne=maxGaussiansPerVoxel-1;
	float minDist[maxGaussiansPerVoxel];//to mantain distance for each index: since K is very small instead of a priority queue we keep a sorted array
	int indAux[maxGaussiansPerVoxel];
	float queryAux[dimsImage];//TODO: I can probably hardcode dimsImage to improve performance (unroll loops)
	float minDistThr;
	float scaleAux[dimsImage];
	scaleAux[0]=knn_v1_scaleCUDA[0];scaleAux[1]=knn_v1_scaleCUDA[1];scaleAux[2]=knn_v1_scaleCUDA[2];

	float dist,distAux;
	int jj2;

	jj2=tid;
	/*texture mmemory
		queryAux[0]=tex1Dfetch(queryTexture,jj2);//stores query point to compare against all the references
		queryAux[1]=tex1Dfetch(queryTexture,jj2+1);
		queryAux[2]=tex1Dfetch(queryTexture,jj2+2);
	 */

	//global memory: organized as x_1,x_2,x_3,....,y_1,y_2,...,z_1,... to have coalescent access
	queryAux[0]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[1]=queryCUDA[jj2];
	jj2+=query_nb;
	queryAux[2]=queryCUDA[jj2];

	int refIdx=-3;
	for(int jj=0;jj<maxGaussiansPerVoxel;jj++) minDist[jj]=1e32;//equivalent to infinity
	minDistThr=minDist[kMinusOne];
	for(int ii=0;ii<ref_nb;ii++)
	{
		__syncthreads();//to access constant memory coherently
		refIdx+=3;
		/*
			dist=0;
			for(int jj=0;jj<dimsImage;jj++) 
			{
				dist+=(queryAux[jj]-knn_v1_refCUDA[refIdx])*(queryAux[jj]-knn_v1_refCUDA[refIdx]);
				refIdx++;
			}
		 */

		distAux=(queryAux[0]-knn_v1_refCUDA[refIdx])*scaleAux[0];
		dist=distAux*distAux;
		if(dist>minDistThr) continue;
		distAux=(queryAux[1]-knn_v1_refCUDA[refIdx+1])*scaleAux[1];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;
		distAux=(queryAux[2]-knn_v1_refCUDA[refIdx+2])*scaleAux[2];
		dist+=distAux*distAux;
		if(dist>minDistThr) continue;

		//decide weather to insert this index or not
		for(jj2=kMinusOne-1;jj2>=0;jj2--)
		{
			if(dist>=minDist[jj2])
			{
				minDist[jj2+1]=dist;
				indAux[jj2+1]=ii;
				break;
			}
			minDist[jj2+1]=minDist[jj2];
			indAux[jj2+1]=indAux[jj2];
		}
		if(jj2==-1)//we need to insert the element at position zero
		{
			minDist[0]=dist;
			indAux[0]=ii;
		}
		minDistThr=minDist[kMinusOne];
	}

	__syncthreads();//I need this to have coalescent memory access to inCUDA: speeds up the code by x4

	//copy indexes to global memory
	jj2=tid;
	for(int jj=0;jj<maxGaussiansPerVoxel;jj++)
	{
		//indCUDA[jj+jj2]=indAux[jj];
		indCUDA[jj2]=indAux[jj];
		jj2+=query_nb;
	}
	//update pointer for next query_point to check
	//tid+=offset;

}


#if 0
//===================================================================================================
int allocateGPUMemoryForKnnCUDA(float *queryTemp,float **queryCUDA,int **indCUDA,long long int query_nb,float *scale)
{
	
	
	HANDLE_ERROR( hipMalloc( (void**)&(*indCUDA), query_nb*maxGaussiansPerVoxel*sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&(*queryCUDA), query_nb*dimsImage*sizeof(float) ) );
	// Copy image data to array
	HANDLE_ERROR(hipMemcpy((*queryCUDA),queryTemp, dimsImage*query_nb*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v1_scaleCUDA),scale, dimsImage * sizeof(float)));//constant memory
	return 0;
}
#endif

#if 0
void setDeviceCUDA(int devCUDA)
{
//WE ASSUME qeuryCUDA AND indCUDA HAVE BEEN ALLOCATED ALREADY AND MEMORY TRANSFERRED TO THE GPU
	HANDLE_ERROR( hipSetDevice( devCUDA ) );
}
#endif

#if 0
//===================================================================================================
void knnCUDA(int *ind,int *indCUDA,float *queryCUDA,float *refTemp,long long int query_nb,int ref_nb)
{


	if(MAX_REF_POINTS<ref_nb)
	{
		//TODO allow th epossibility of more ref_points by using global memory instead of constant memory
		printf("ERROR!! Increase MAX_REF_POINTS!\n");
		exit(2);
	}
	if(dimsImage!=3)
	{
		printf("ERROR: dimsImage should be 3\n");
		exit(2);
	}
	//calculate number of threads and blocks
long long 	int numThreads=std::min((long long int)MAX_THREADS,query_nb);
long long 	int numGrids=std::min((long long int)MAX_BLOCKS,(query_nb+numThreads-1)/numThreads);
	
	//printf("NumThreads=%d;numGrids=%d\n",numThreads,numGrids);
	
	
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v1_refCUDA),refTemp,ref_nb* dimsImage * sizeof(float)));//constant memory	
	knnKernel<<<numGrids,numThreads>>>(indCUDA,queryCUDA,ref_nb,query_nb);
	HANDLE_ERROR_KERNEL;


	HANDLE_ERROR(hipMemcpy(ind,indCUDA,query_nb*maxGaussiansPerVoxel*sizeof(int),hipMemcpyDeviceToHost));//retrieve indexes: memcopy is synchronous unless stated otherwise

}
#endif 

//===================================================================================================
void knn_v1_CUDAinPlace(int *indCUDA,float *queryCUDA,float *refTemp,long long int query_nb,int ref_nb)
{

	
	if(dimsImage!=3)
	{
		printf("ERROR: dimsImage should be 3\n");
		exit(2);
	}
	//calculate number of threads and blocks
long long 	int numThreads=std::min((long long int)MAX_THREADS,query_nb);
long long 	int numGrids=std::min((long long int)MAX_BLOCKS,(query_nb+numThreads-1)/numThreads);
	
	if(MAX_REF_POINTS<ref_nb)//use global memory for anchorPoints (slower)
	{
		knnKernelNoConstantMemory<<<numGrids,numThreads>>>(indCUDA,queryCUDA,refTemp,ref_nb,query_nb);HANDLE_ERROR_KERNEL;	
	}else{//use constant memory for anchor points
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v1_refCUDA),refTemp,ref_nb* dimsImage * sizeof(float),0,hipMemcpyDeviceToDevice));//constant memory	
		knnKernel<<<numGrids,numThreads>>>(indCUDA,queryCUDA,ref_nb,query_nb);HANDLE_ERROR_KERNEL;
	}
}


#if 0
//====================================================================================================
void deallocateGPUMemoryForKnnCUDA(float **queryCUDA,int **indCUDA)
{
	HANDLE_ERROR( hipFree( *indCUDA ) );
	(*indCUDA)=NULL;
    HANDLE_ERROR( hipFree( *queryCUDA ) );
    (*queryCUDA)=NULL;
}
#endif 

//==============================================================
void knn_v1_uploadScaleCUDA(float *scale)
{
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(knn_v1_scaleCUDA),scale, dimsImage * sizeof(float)));//constant memory
}
